#include "head.h"
float htd(dt *x,long a,long b,long c,int *k,dt rel_eps,dt max_rank)
{	
	
	float time_elapsed;
	hipEvent_t start,stop;
	hipEventCreate(&start);       //创建Event
	hipEventCreate(&stop);
	hipEventRecord(start,0);    //记录当前时间

	dt *d_X,*d_B1,*d_B2;
	hipMalloc((void**)&d_X,sizeof(dt)*a*b*c);  //原 x，也是mode-1 的x
	
	hipMemcpy(d_X,x,sizeof(dt)*a*b*c,hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	dim3 threads(1024,1,1);
	dim3 block0((a*b*c+1024-1)/1024,1,1); // mode-2



	//Node2 qr_svd
	dt *d_X_node2;
	hipMalloc((void**)&d_X_node2,sizeof(dt)*a*b*c);
	tensorToMode1<<<block0,threads>>>(d_X,d_X_node2,a,b,c);
	//hipMemcpy(d_X_node2,x,sizeof(dt)*a*b*c,hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	
	dt *d_X2,*d_X3,*d_X1;
	hipMalloc((void**)&d_X2,sizeof(dt)*a*b*c);
	hipMalloc((void**)&d_X1,sizeof(dt)*a*b*c);
	hipMalloc((void**)&d_X3,sizeof(dt)*a*b*c);
	hipDeviceSynchronize();
	// 函数准备
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	dt alpha = 1.0;
	dt beta = 0.0;
	dt alpha1=-1.0;
	dt re=0.0;
	dt before = 0.0;

	dt *d_X1_X1,*d_X2_X2,*d_X3_X3;
	hipMalloc((void**)&d_X1_X1,sizeof(dt)*a*a);
	hipMalloc((void**)&d_X2_X2,sizeof(dt)*b*b);
	hipMalloc((void**)&d_X3_X3,sizeof(dt)*c*c);

	dt *d_Ux5,*d_Ux4,*d_Ux3,*d_Ux2;
	hipMalloc((void**)&d_Ux5,sizeof(dt)*b*k[4]);
	hipMalloc((void**)&d_Ux4,sizeof(dt)*a*k[3]);
	hipMalloc((void**)&d_Ux3,sizeof(dt)*c*k[2]);
	hipMalloc((void**)&d_Ux2,sizeof(dt)*a*b*k[1]);
	tensorToMode1<<<block0,threads>>>(d_X,d_X1,a,b,c);
	hipDeviceSynchronize();

	tensorToMode2<<<block0,threads>>>(d_X,d_X2,a,b,c);
	hipDeviceSynchronize();
	tensorToMode3<<<block0,threads>>>(d_X,d_X3,a,b,c);
	hipDeviceSynchronize();

	/*
	*   QR-SVD
	*/
	qr_svd_2(d_X1,d_X1_X1,a,b*c);

	//svd_VT(d_X1,d_X1_X1,a,b*c,handle);
	transmission<<<block0,threads>>>(d_X1_X1,d_Ux4,a,k[3]);
	hipDeviceSynchronize();
	

	qr_svd_2(d_X2,d_X2_X2,b,a*c);
	transmission<<<block0,threads>>>(d_X2_X2,d_Ux5,b,k[4]);
	hipDeviceSynchronize();
	qr_svd_2(d_X3,d_X3_X3,c,b*a);
	transmission<<<block0,threads>>>(d_X3_X3,d_Ux3,c,k[2]);
	hipDeviceSynchronize();

		
	//Node 2 non-leaf mode-(12)=mode3T, svd->ttm->B{2}

	dt *d_U;
	hipMalloc((void**)&d_U,sizeof(dt)*a*b*c);

	hipDeviceSynchronize();
	qr_svd(d_X_node2,d_U,a*b,c); 
	//gesvda(d_X_node2,d_U,a*b,c,20);
	transmission<<<block0,threads>>>(d_U,d_Ux2,a*b,k[1]);
	hipDeviceSynchronize();
	float *d_Ux2_tensor;
	hipMalloc((void**)&d_Ux2_tensor,sizeof(float)*a*b*k[1]);
	tensorToMode1<<<block0,threads>>>(d_Ux2,d_Ux2_tensor,a,b,k[1]);
	//cout<<"--------"<<endl;printTensor(d_Ux2,4,4,1);            


	//U{2}还原张量后X 与 U{4} U{5} ttm
	//(1) mode-3 转置求mode-1
	
	//  ttm(U{2}的tensor X1 U{4} X2 U{5})    
	dt *d_XU4,*d_XU2,*d_XU4_2,*d_B2_2,*d_XU2_2,*d_B1_2;
	hipMalloc((void**)&d_XU4,sizeof(dt)*k[3]*b*k[1]);
	hipMalloc((void**)&d_XU4_2,sizeof(dt)*k[3]*b*k[1]);
	hipMalloc((void**)&d_XU2,sizeof(dt)*k[1]*c);
	hipMalloc((void**)&d_XU2_2,sizeof(dt)*k[1]*c);
	hipMalloc((void**)&d_B1,sizeof(dt)*k[1]*k[2]); 
	hipMalloc((void**)&d_B1_2,sizeof(dt)*k[1]*k[2]); 
	hipMalloc((void**)&d_B2,sizeof(dt)*k[3]*k[4]*k[1]);
	hipMalloc((void**)&d_B2_2,sizeof(dt)*k[3]*k[4]*k[1]);
	hipDeviceSynchronize();
	tensorToMode1<<<block0,threads>>>(d_Ux2_tensor,d_Ux2,a,b,k[1]);
	/*
	*   float to half 的变量
	*/
	half *d_Ux4_h, *d_Ux2_h, *d_Ux5_h, *d_XU4_h, *d_X_h, *d_XU2_h, *d_Ux3_h;
	hipMalloc((void**)&d_Ux4_h,sizeof(half)*a*k[3]);
	hipMalloc((void**)&d_Ux2_h,sizeof(half)*a*b*k[1]);
	hipMalloc((void**)&d_Ux5_h,sizeof(half)*b*k[4]);
	hipMalloc((void**)&d_XU4_h,sizeof(half)*k[3]*b*k[1]);
	hipMalloc((void**)&d_X_h,sizeof(half)*a*b*c);
	hipMalloc((void**)&d_XU2_h,sizeof(half)*k[1]*c*1);	
	hipMalloc((void**)&d_Ux3_h,sizeof(half)*c*k[2]);
	
	//hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,k[3],b*k[1],a,&alpha,d_Ux4,a,d_Ux2,a,&beta,d_XU4_2,k[2]);
	f2h(d_Ux4,d_Ux4_h,a*k[3]);
	f2h(d_Ux2,d_Ux2_h,a*b*k[1]);
	hipblasGemmEx(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,k[3],b*k[1],a,
		&alpha,d_Ux4_h,HIP_R_16F,a,
		d_Ux2_h,HIP_R_16F,a,
		&beta,d_XU4_2,HIP_R_32F,k[2],
		HIP_R_32F,CUBLAS_GEMM_DEFAULT_TENSOR_OP);


	float *d_XU4_tensor;
	hipMalloc((void**)&d_XU4_tensor,sizeof(dt)*k[3]*b*k[1]);
	/*hipblasSgemmStridedBatched(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,k[3],b,a,
	                          &alpha,d_Ux4,a,0,d_Ux2,a,a*b,&beta,d_XU4_2,k[3],k[3]*b,k[1]
	                          );*/
	tensorToMode1<<<block0,threads>>>(d_XU4_2,d_XU4,k[3],b,k[1]);
	hipDeviceSynchronize();
	tensorToMode2<<<block0,threads>>>(d_XU4,d_XU4_tensor,k[3],b,k[1]);
	hipDeviceSynchronize();


	//hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,k[4],k[3]*k[1],b,&alpha,d_Ux5,b,d_XU4_tensor,b,&beta,d_B2_2,k[4]);
	f2h(d_Ux5,d_Ux5_h,b*k[4]);
	f2h(d_XU4_tensor,d_XU4_h,k[3]*b*k[1]);
	hipblasGemmEx(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,k[4],k[3]*k[1],b,
		&alpha,d_Ux5_h,HIP_R_16F,b,
		d_XU4_h,HIP_R_16F,b,
		&beta,d_B2_2,HIP_R_32F,k[4],
		HIP_R_32F,CUBLAS_GEMM_DEFAULT_TENSOR_OP);

	tensorToMode2<<<block0,threads>>>(d_B2_2,d_B2,k[4],k[3],k[1]);                                             
	hipDeviceSynchronize();


	//Node Root U{1}=vec(x),这里张量化后结果为直接对X 取数 (a*b)*c
	//然后 对 U2 U3 ttm （U2 9*3 ，U3 3*3 ）->B1 3*3
	float *d_Xmode3;
	hipMalloc((void**)&d_Xmode3,sizeof(float)*a*b*c);
	tensorToMode2<<<block0,threads>>>(d_X,d_Xmode3,a,b,c);
	hipDeviceSynchronize();
	//hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,k[1],c,a*b,&alpha,d_Ux2,a*b,d_X,a*b,&beta,d_XU2_2,k[1]);
	f2h(d_X,d_X_h,a*b*c);
	hipblasGemmEx(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,k[1],c,a*b,
		&alpha,d_Ux2_h,HIP_R_16F,a*b,
		d_X_h,HIP_R_16F,a*b,
		&beta,d_XU2_2,HIP_R_32F,k[1],
		HIP_R_32F,CUBLAS_GEMM_DEFAULT_TENSOR_OP);

	float *d_XU2_tensor,*d_B1_tensor;
	hipMalloc((void**)&d_XU2_tensor,sizeof(float)*k[1]*c*1);
	hipMalloc((void**)&d_B1_tensor,sizeof(float)*k[1]*k[2]);
	tensorToMode2<<<block0,threads>>>(d_XU2_2,d_XU2,k[1],c,1);
	tensorToMode2<<<block0,threads>>>(d_XU2,d_XU2_tensor,k[1],c,1);
	hipDeviceSynchronize();

	//hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,k[2],k[1]*1,c,&alpha,d_Ux3,c,d_XU2,c,&beta,d_B1_2,k[2]);
	f2h(d_XU2,d_XU2_h,k[1]*c*1);
	f2h(d_Ux3,d_Ux3_h,c*k[2]);
	hipblasGemmEx(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,k[2],k[1]*1,c,
		&alpha,d_Ux3_h,HIP_R_16F,c,
		d_XU2_h,HIP_R_16F,c,
		&beta,d_B1_2,HIP_R_32F,k[2],
		HIP_R_32F,CUBLAS_GEMM_DEFAULT_TENSOR_OP);	


	tensorToMode2<<<block0,threads>>>(d_B1_2,d_B1,k[2],k[1],1);
	tensorToMode2<<<block0,threads>>>(d_B1,d_B1,k[2],k[1],1);
	hipDeviceSynchronize();
	hipFree(d_B1_2);
	hipFree(d_XU2_2);
	hipFree(d_B2_2);
	hipFree(d_XU4_2);		
	hipFree(d_Ux2_tensor);
	hipFree(d_XU2_tensor);
	hipFree(d_B1_tensor);
	hipFree(d_XU4_tensor);


	hipEventRecord(stop,0);    //记录当前时间
	hipEventSynchronize(start);    //Waits for an event to complete.
	hipEventSynchronize(stop);    //Waits for an event to complete.Record之前的任务
	hipEventElapsedTime(&time_elapsed,start,stop);    //计算时间差
	hipEventDestroy(start);    //destory the event
	hipEventDestroy(stop);
	time_elapsed = time_elapsed/1000;
	cout<<"cost time :"<<time_elapsed<<"s"<<endl;

//finish decomposition B{1}->d_B1,B{2}->d_B2,U{3}->d_X3X3,U{4}->d_X1X1,U{5}->d_X2X2
//recover the tensor x
//ttm(B{2},U{4},U{5})-->U{2}
	dt *d_U4B2,*d_U2B1,*d_r,*d_U4B2_2,*d_U2B1_2,*d_r_2;
	hipMalloc((void**)&d_U4B2,sizeof(dt)*a*k[4]*k[1]);
	hipMalloc((void**)&d_U4B2_2,sizeof(dt)*a*k[4]*k[1]);
	hipMalloc((void**)&d_U2B1,sizeof(dt)*a*b*k[2]);
	hipMalloc((void**)&d_U2B1_2,sizeof(dt)*a*b*k[2]);
	hipMalloc((void**)&d_r,sizeof(dt)*a*b*c);
	hipMalloc((void**)&d_r_2,sizeof(dt)*a*b*c);
	dt *d_U2_r,*d_U2_r_2;
	hipMalloc((void**)&d_U2_r,sizeof(dt)*a*b*k[1]);
	hipMalloc((void**)&d_U2_r_2,sizeof(dt)*a*b*k[1]);
	hipDeviceSynchronize();

hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,a,k[4],k[3],
                          &alpha,d_Ux4,a,0,d_B2,k[3],k[3]*k[4],
                          &beta,d_U4B2_2,a,a*k[4],k[1]
                          );
tensorToMode2<<<block0,threads>>>(d_U4B2_2,d_U4B2,a,k[4],k[1]);

    hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,b,a,k[4],
    						&alpha,d_Ux5,b,0,d_U4B2,k[4],k[4]*a,
    						&beta,d_U2_r_2,b,b*a,k[1]
    						); 
    tensorToMode2<<<block0,threads>>>(d_U2_r_2,d_U2_r,b,a,k[1]);					                         
    hipDeviceSynchronize();
  
//ttm(B{1},U{2},U{3})-->U{1}  

  hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,a*b,k[2],k[1],
                            &alpha,d_U2_r,a*b,0,d_B1,k[1],k[1]*k[2],
                            &beta,d_U2B1_2,a*b,a*b*k[2],1
                            );
  tensorToMode2<<<block0,threads>>>(d_U2B1_2,d_U2B1,a*b,k[2],1);
  hipDeviceSynchronize();
 
    hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,
                              c,a*b,k[2],
                              &alpha,d_Ux3,c,0,d_U2B1,k[2],k[2]*a*b,
                              &beta,d_r_2,c,c*a*b,1
                              );
    tensorToMode2<<<block0,threads>>>(d_r_2,d_r,c,a*b,1);
    hipDeviceSynchronize();
    hipFree(d_U2_r_2);
    hipFree(d_r_2);
    hipFree(d_U4B2_2);
    hipFree(d_U2B1_2);
    // cout<<"recover------"<<endl;printTensor(d_r,6,3,1);
    //cout<<"original"<<endl;printTensor(d_X,3,3,1);
	
	//compute error		
	





	//d_r=-d_X + d_r
	hipblasSaxpy(handle,a*b*c,&alpha1,d_X,1,d_r,1); 
	hipDeviceSynchronize();

	hipblasSnrm2(handle,a*b*c,d_r,1,&re);
	hipblasSnrm2(handle,a*b*c,d_X,1,&before);


	hipDeviceSynchronize();
	cout<<"error rate "<<re/before<<endl;
	
	ofstream fout("time.txt",ios::app);
	fout<<time_elapsed<<"  "<<re/before<<endl;
	fout.close();


	hipFree(d_X);

	hipFree(d_Ux3);
	hipFree(d_Ux2);
	hipFree(d_XU4);
	hipFree(d_Ux5);
	hipFree(d_Ux4);


	hipFree(d_XU4);
	hipFree(d_XU2);
	hipFree(d_B2);
	hipFree(d_B1);

	hipFree(d_U4B2);
	hipFree(d_U2B1);
	hipFree(d_X2_X2);
	hipFree(d_X1_X1);
	hipFree(d_X3_X3);
	hipFree(d_U);
	hipblasDestroy(handle);

	return re/before;

}