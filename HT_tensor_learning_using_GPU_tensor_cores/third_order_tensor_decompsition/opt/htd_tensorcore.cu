#include "head.h"
float htd(dt *x,long a,long b,long c,int *k)
{		

	float time_elapsed;
	hipEvent_t start,stop;
	hipEventCreate(&start);       //创建Event
	hipEventCreate(&stop);
	hipEventRecord(start,0);    //记录当前时间

	dt *d_X;
	hipMalloc((void**)&d_X,sizeof(dt)*a*b*c);  //原 x，也是mode-1 的x	
	hipMemcpy(d_X,x,sizeof(dt)*a*b*c,hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	//cout<<"start~~~~"<<endl;printTensor(d_X,4,4,1);


	half *d_X_h;
	hipMalloc((void**)&d_X_h,sizeof(half)*a*b*c);
	f2h(d_X,d_X_h,a*b*c);


	//Node2
	dt *d_X_node2;
	hipMalloc((void**)&d_X_node2,sizeof(dt)*a*b*c);
	hipMemcpy(d_X_node2,x,sizeof(dt)*a*b*c,hipMemcpyHostToDevice);

	//half *d_X_node2_h;
	//hipMalloc((void**)&d_X_node2_h,sizeof(half)*a*b*c);
	//f2h(d_X_node2,d_X_node2_h,a*b*c);


	dt *d_X2;
	hipMalloc((void**)&d_X2,sizeof(dt)*a*b*c);
	half *d_X2_h;
	hipMalloc((void**)&d_X2_h,sizeof(half)*a*b*c);

	hipDeviceSynchronize();
	// For function
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSetMathMode(handle,HIPBLAS_TENSOR_OP_MATH);
	dt alpha = 1.0;
	dt beta = 0.0;
	dt alpha1=-1.0;
	dt re=0.0;
	dt before = 0.0;

	hipsolverHandle_t cusolverH = NULL;

  	hipStream_t stream = NULL;
 	hipsolverSyevjInfo_t syevj_params = NULL;


	dim3 threads(1024,1,1);
	dim3 block0((a*b*c+1024-1)/1024,1,1); // for mode-2
	hipDeviceSynchronize();
	dt *d_X1_X1,*d_X2_X2,*d_X3_X3;
	hipMalloc((void**)&d_X1_X1,sizeof(dt)*a*a);
	hipMalloc((void**)&d_X2_X2,sizeof(dt)*b*b);
	hipMalloc((void**)&d_X3_X3,sizeof(dt)*c*c);

	dt *d_Ux5,*d_Ux4,*d_Ux3,*d_Ux2;
	hipMalloc((void**)&d_Ux5,sizeof(dt)*b*k[4]);
	hipMalloc((void**)&d_Ux4,sizeof(dt)*a*k[3]);
	hipMalloc((void**)&d_Ux3,sizeof(dt)*c*k[2]);
	hipMalloc((void**)&d_Ux2,sizeof(dt)*a*b*k[1]);

	half *d_Ux5_h,*d_Ux4_h,*d_Ux3_h,*d_Ux2_h;
	hipMalloc((void**)&d_Ux5_h,sizeof(half)*b*k[4]);
	hipMalloc((void**)&d_Ux4_h,sizeof(half)*a*k[3]);
	hipMalloc((void**)&d_Ux3_h,sizeof(half)*c*k[2]);
	hipMalloc((void**)&d_Ux2_h,sizeof(half)*a*b*k[1]);


	hipDeviceSynchronize();

	//1、mode-2
	//mode2<<<block0,threads>>>(d_X,d_X2,a,b,c);
	mode2h<<<block0,threads>>>(d_X_h,d_X2_h,a,b,c);
	hipDeviceSynchronize();
	// mode-3  d_x3
	hipblasGemmEx(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,
                           b,b,a*c,
                           &alpha,d_X2_h,HIP_R_16F,b,
                           d_X2_h,HIP_R_16F,b,
                           &beta,d_X2_X2,HIP_R_32F,b,
                           HIP_R_32F,
                           CUBLAS_GEMM_DEFAULT_TENSOR_OP);
	//printTensor(d_X2_X2,4,4,1);
	/*hipblasSgemm(handle,
	            HIPBLAS_OP_N,
	            HIPBLAS_OP_T,
	            b,b,a*c,
	            &alpha,d_X2,b,d_X2,b,
	            &beta,d_X2_X2,b
	            );*/
	hipblasGemmEx(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,
                           a,a,b*c,
                           &alpha,d_X_h,HIP_R_16F,a,
                           d_X_h,HIP_R_16F,a,
                           &beta,d_X1_X1,HIP_R_32F,a,
                           HIP_R_32F,
                           CUBLAS_GEMM_DEFAULT_TENSOR_OP);

	/*hipblasSgemm(handle,
	            HIPBLAS_OP_N,
	            HIPBLAS_OP_T,
	            a,a,b*c,
	            &alpha,d_X,a,d_X,a,
	            &beta,d_X1_X1,a
	            );*/
	hipDeviceSynchronize();

	hipblasGemmEx(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,
                           c,c,a*b,
                           &alpha,d_X_h,HIP_R_16F,a*b,
                           d_X_h,HIP_R_16F,a*b,
                           &beta,d_X3_X3,HIP_R_32F,c,
                           HIP_R_32F,
                           CUBLAS_GEMM_DEFAULT_TENSOR_OP);

	/*hipblasSgemm(handle,
	            HIPBLAS_OP_T,
	            HIPBLAS_OP_N,
	            c,c,a*b,
	            &alpha,d_X,a*b,d_X,a*b,
	            &beta,d_X3_X3,c
	            );*/
	hipDeviceSynchronize();

	hipsolverDnCreate(&cusolverH);
	evd(d_X1_X1,a,handle,cusolverH);	
	hipblasScopy(handle,a*k[3],d_X1_X1+a*(a-k[3]) ,1,d_Ux4,1);

	evd(d_X2_X2,b,handle,cusolverH);
	hipblasScopy(handle,b*k[4],d_X2_X2+b*(b-k[4]) ,1,d_Ux5,1);
	
	evd(d_X3_X3,c,handle,cusolverH);
	hipblasScopy(handle,c*k[2],d_X3_X3+c*(c-k[2]) ,1,d_Ux3,1);




	/*dt *d_sumXXT;
	hipMalloc((void**)&d_sumXXT,sizeof(dt)*a*a*3);


	hipblasScopy(handle,a*a,d_X1_X1 ,1,d_sumXXT,1);
	hipblasScopy(handle,b*b,d_X2_X2 ,1,d_sumXXT+a*a,1);
	hipblasScopy(handle,c*c,d_X3_X3 ,1,d_sumXXT+a*a+b*b,1);

	
	dt *d_W = NULL; 
	int* d_info = NULL;
	int lwork = 0; 
	dt *d_work = NULL;

const dt tol = 1.e-6;
const int max_sweeps = 500;
const int sort_eig = 1; 
const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; 
const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
hipsolverDnCreate(&cusolverH);
hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
hipsolverSetStream(cusolverH, stream);
hipsolverDnCreateSyevjInfo(&syevj_params);
hipsolverDnXsyevjSetTolerance(syevj_params,tol);
hipsolverDnXsyevjSetMaxSweeps(syevj_params,max_sweeps);
hipsolverDnXsyevjSetSortEig(syevj_params,sort_eig);
hipMalloc ((void**)&d_W , sizeof(dt) * a * 3);
hipMalloc ((void**)&d_info, sizeof(int ) * 3);
 hipsolverDnSsyevjBatched_bufferSize(
         cusolverH,
         jobz,
         uplo,
         a,
          d_sumXXT,
         a,
          d_W,
          &lwork,
          syevj_params,
          3
          );
  hipMalloc((void**)&d_work, sizeof(dt)*lwork);
 
  hipsolverDnSsyevjBatched(
        cusolverH,
        jobz,
        uplo,
        a,
        d_sumXXT,
        a,
        d_W,
        d_work,
        lwork,
        d_info,
        syevj_params,
        3
        );
	hipDeviceSynchronize();

	int info;
	hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);

	if ( 0 == info ){
	printf(" converges \n");
	}else if ( 0 > info ){
	printf("%d-th parameter is wrong \n", -info);
	exit(1);
	}else{
	printf("WARNING: info = %d :  does not converge \n", info );
	}
	//ascending order
	hipDeviceSynchronize();

	//tr<<<bl2,th2>>>(d_Ux4,d_Ux5,d_Ux3,d_sumXXT,a,k[3],k[4],k[2]);
	//hipDeviceSynchronize();
	//printTensor(d_Ux4,5,5,1);

	hipblasScopy(handle,a*k[3],d_sumXXT+a*(a-k[3]) ,1,d_Ux4,1);
	hipblasScopy(handle,b*k[4],d_sumXXT+a*a+a*(a-k[4]) ,1,d_Ux5,1);
	hipblasScopy(handle,c*k[2],d_sumXXT+2*a*a+a*(a-k[2]) ,1,d_Ux3,1);*/
	
	//d_Ux4=d_sumXXT+a*(a-k[3]);
	//d_Ux5=d_sumXXT+a*a+a*(a-k[4]);
	//d_Ux3=d_sumXXT+2*a*a+a*(a-k[2]);

	//Node 2 non-leaf mode-(12)=mode3T, svd->ttm->B{2}

	/*dt *d_U,*d_Ux2_t;
	hipMalloc((void**)&d_U,sizeof(dt)*a*b*c);
	hipMalloc((void**)&d_Ux2_t,sizeof(dt)*a*b*c);
	gesvda(d_X_node2,d_U,a*b,c,k[1]); // 降序

	transmission<<<block0,threads>>>(d_U,d_Ux2_t,a*b,c);
	hipDeviceSynchronize();

	//d_Ux2=d_Ux2_t+(b-k[1])*a*b;
	hipblasScopy(handle,a*b*k[1],d_Ux2_t+(b-k[1])*a*b,1,d_Ux2,1);
	hipDeviceSynchronize();*/
	dt *d_U,*d_Ux2_t;
	hipMalloc((void**)&d_U,sizeof(dt)*a*b*k[1]);

	hipsolverDnDestroy(cusolverH);
	hipStreamDestroy(stream);
	hipsolverDnDestroySyevjInfo(syevj_params);
    hipsolverDnCreate(&cusolverH);
	rsvd(d_X_node2,a*b,c,k[1],d_U, handle, cusolverH);

	dim3 threads_2(1024,1,1);
	dim3 block_2((a*b*k[1]+1024-1)/1024,1,1);

	transmission<<<block_2,threads_2>>>(d_U,d_Ux2,a*b,k[1]);
	hipDeviceSynchronize();
	//cout<<"U2 values--------"<<endl;printTensor(d_Ux2,4,4,1);

	dt *d_B2,*d_B1;
	hipMalloc((void**)&d_B1,sizeof(dt)*k[1]*k[2]); 
	hipMalloc((void**)&d_B2,sizeof(dt)*k[3]*k[4]*k[1]);

	f2h(d_Ux2,d_Ux2_h,a*b*k[1]);
	f2h(d_Ux3,d_Ux3_h,c*k[2]);
	f2h(d_Ux4,d_Ux4_h,a*k[3]);
	f2h(d_Ux5,d_Ux5_h,b*k[4]);

	//  ttm(U{2}的tensor X1 U{4} X2 U{5})
	//ttm(d_Ux2,d_Ux4,d_Ux5,d_B2,a,b,k[1],k[3],k[4],handle);
	ttm_tensorcore(d_Ux2_h,d_Ux4_h,d_Ux5_h,d_B2,a,b,k[1],k[3],k[4],handle);


	//ttm(d_X,d_Ux2,d_Ux3,d_B1,a*b,c,k[0],k[1],k[2],handle);
	ttm_tensorcore(d_X_h,d_Ux2_h,d_Ux3_h,d_B1,a*b,c,k[0],k[1],k[2],handle);
	//cout<<"tensor B1:"<<endl;;printTensor(d_B1,5,5,1);

    hipEventRecord( stop,0);    //记录当前时间
	hipEventSynchronize(start);    //Waits for an event to complete.
	hipEventSynchronize(stop);    //Waits for an event to complete.Record之前的任务
	hipEventElapsedTime(&time_elapsed,start,stop);    //计算时间差
	hipEventDestroy(start);    //destory the event
	hipEventDestroy(stop);
	time_elapsed = time_elapsed/1000;
	cout<<"cost time :"<<time_elapsed<<"s"<<endl;




//finish decomposition B{1}->d_B1,B{2}->d_B2,U{3}->d_X3X3,U{4}->d_X1X1,U{5}->d_X2X2
//recover the tensor x
//ttm(B{2},U{4},U{5})-->U{2}
/*****************
*这里使用tensor core计算来还原原始tensor意义不大，时间上加速不多，理论上精度会有一些损失
*
*******************/
	dt *d_U4B2,*d_U2B1,*d_r;
	hipMalloc((void**)&d_U4B2,sizeof(dt)*a*k[4]*k[1]);
	hipMalloc((void**)&d_U2B1,sizeof(dt)*a*b*k[2]);
	hipMalloc((void**)&d_r,sizeof(dt)*a*b*c);
	dt *d_U2_r;
	hipMalloc((void**)&d_U2_r,sizeof(dt)*a*b*k[1]);
	hipDeviceSynchronize();

	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,
	            a,k[4]*k[1],k[3],
	            &alpha,d_Ux4,a,d_B2,k[3],
	            &beta,d_U4B2,a
      	      	);

    hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,
                              a,b,k[4],
                              &alpha,d_U4B2,a,a*k[4],d_Ux5,b,0,
                              &beta,d_U2_r,a,a*b,k[1]
                              );
    hipDeviceSynchronize();

    //printTensor(d_U2,6,6,1);
//ttm(B{1},U{2},U{3})-->U{1}  

    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,
                a*b,k[2],k[1],
                &alpha,d_U2_r,a*b,d_B1,k[1],
                &beta,d_U2B1,a*b
                );
    /*cout<<"zhong jian d_U2_r------"<<endl;printTensor(d_U2_r,4,4,1);
    cout<<"zhong jian d_B1------"<<endl;printTensor(d_B1,4,4,1);
    cout<<"zhong jian d_U2B1------"<<endl;printTensor(d_U2B1,4,4,1);*/
    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,
                a*b,c,k[2],
                &alpha,d_U2B1,a*b,d_Ux3,c,
                &beta,d_r,a*b
                );

    hipDeviceSynchronize();
    //cout<<"recover------"<<endl;printTensor(d_r,6,3,1);
    //cout<<"original"<<endl;printTensor(d_X,3,3,1);
	

    
    

	
	//compute error		
	//d_r=-d_X + d_r
	hipblasSaxpy(handle,a*b*c,&alpha1,d_X,1,d_r,1); 
	hipDeviceSynchronize();

	hipblasSnrm2(handle,a*b*c,d_r,1,&re);
	hipblasSnrm2(handle,a*b*c,d_X,1,&before);
	hipDeviceSynchronize();
	
	cout<<"error rate "<<re/before<<endl;
	
	ofstream fout("time.txt",ios::app);
	fout<<time_elapsed<<"  "<<re/before<<endl;
	fout.close();





	//hipFree(d_W);
	//hipFree(d_work);
	hipFree(d_X);
	hipFree(d_X2);
	hipFree(d_X1_X1);
	hipFree(d_X2_X2);
	hipFree(d_X3_X3);	
	hipFree(d_Ux3);
	hipFree(d_Ux2);
	//hipFree(d_XU4);
	hipFree(d_Ux5);
	//hipFree(d_XU4);
	//hipFree(d_XU2);
	hipFree(d_B2);
	hipFree(d_B1);
	hipFree(d_U4B2);
	hipFree(d_U2B1);
	hipFree(d_X2_X2);
	hipFree(d_X1_X1);
	hipFree(d_X3_X3);
	//hipFree(d_U);
	hipblasDestroy(handle);
	hipsolverDnDestroy(cusolverH);
	//hipStreamDestroy(stream);
	//hipsolverDnDestroySyevjInfo(syevj_params);

	return re/before;

}