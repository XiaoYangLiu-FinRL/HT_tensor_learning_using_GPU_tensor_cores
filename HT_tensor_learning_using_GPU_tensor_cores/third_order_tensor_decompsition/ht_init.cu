#include "head.h"
#include<unistd.h>
#include "nvml.h"

int main()
{

	
	for(long j=1120;j<=1140;j=j+80){
		long a=j; //size x
		long b=j;
		long c=j;
		
		cout<<"size:"<<j<<endl;
		
		int *k=new int[j]();
		for(int i=1;i<5;i++)
			k[i]=(int)(j*0.1);
			//k[i]=j;
		k[0]=1;
		//k[1]=2;

		dt *X;
		float error;
		hipHostAlloc((void**)&X,sizeof(dt)*a*b*c,0);
		genHtensor(X,a,b,c); //init tensor
		//gentuTensor1(X,a,b,c,k[1],k[2],k[3]);
		nvmlReturn_t result;
		result = nvmlInit();
		int device_count;
		hipGetDevice(&device_count);
		

		nvmlDevice_t device;
        char name[NVML_DEVICE_NAME_BUFFER_SIZE];
        nvmlPciInfo_t pci;
        result = nvmlDeviceGetHandleByIndex(device_count, &device);
        if (NVML_SUCCESS != result) {
            std::cout << "get device failed " << endl;
        }
        result = nvmlDeviceGetName(device, name, NVML_DEVICE_NAME_BUFFER_SIZE);
        if (NVML_SUCCESS != result) {
            std::cout << "GPU name： " << name << endl;
        }
        

		error = htd(X,a,b,c,k);

		    //使用率
        nvmlUtilization_t utilization;
        result = nvmlDeviceGetUtilizationRates(device, &utilization);
        if (NVML_SUCCESS == result)
        {
            std::cout << " device :"<< device_count <<"utilize.";
            std::cout << " GPU utilize： " << utilization.gpu << " device memory utilize " << utilization.memory << endl;
        }else{
        	cout<<"fail ！"<<endl;
        }
		



		//htd_tensor(X,a,b,c,k);

		hipHostFree(X);
		hipDeviceReset();
}
	return 0;

}