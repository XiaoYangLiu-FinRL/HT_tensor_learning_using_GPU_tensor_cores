#include "head.h"
void htd(dt *X,long a,long b,long c,int *k)
{


	float time_elapsed;
	hipEvent_t start,stop;
	hipEventCreate(&start);       //创建Event
	hipEventCreate(&stop);
	hipEventRecord(start,0);    //记录当前时间



	int p,slice;  // each part process slice matrices, leave le matrix
	if(c%4==0){
		p = 4;   //4 parts
		slice = c/4; 
	}
	dim3 threads0(512,1,1);
	dim3 block00((a*a+512-1)/512,1,1); //for X2
	dim3 block1((slice*slice+512-1)/512,1,1); //for X3
	dt alpha = 1.0;
	dt beta = 0.0;
	dt beta1=1.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle); 	
	hipsolverHandle_t cusolverH = NULL;
	hipsolverDnCreate(&cusolverH);
	hipblasSetMathMode(handle,HIPBLAS_TENSOR_OP_MATH);

	dt *d_X1_X1,*d_X2_X2,*d_X3_X3;
	hipMalloc((void**)&d_X1_X1,sizeof(dt)*a*a);
	hipMalloc((void**)&d_X2_X2,sizeof(dt)*b*b);
	hipMalloc((void**)&d_X3_X3,sizeof(dt)*c*c);

	dt *d_X2,*d_X3,*d_X3T,*d_Idemat3,*d_Idemat;	
	hipMalloc((void**)&d_X2,sizeof(dt)*a*b*slice);
	hipMalloc((void**)&d_Idemat,sizeof(dt)*a*a);
	hipMalloc((void**)&d_X3,sizeof(dt)*c*c);
	hipMalloc((void**)&d_X3T,sizeof(dt)*c*slice);
	hipMalloc((void**)&d_Idemat3,sizeof(dt)*slice*slice);
	initIdeMat<<<block1,threads0>>>(d_Idemat3,slice);
	initIdeMat<<<block00,threads0>>>(d_Idemat,a);


	half *h_Idemat,*h_X2;
	hipMalloc((void**)&h_Idemat,sizeof(half)*a*a);
	hipMalloc((void**)&h_X2,sizeof(half)*a*b*slice);
	f2h(d_Idemat,h_Idemat,a*a);

	dt *d_Xtemp,*d_Xtemp1;
	hipMalloc((void**)&d_Xtemp,sizeof(dt)*a*b*slice);
	hipMalloc((void**)&d_Xtemp1,sizeof(dt)*a*b*slice);
	half *h_Xtemp,*h_Xtemp1;
	hipMalloc((void**)&h_Xtemp,sizeof(half)*a*b*slice);
	hipMalloc((void**)&h_Xtemp1,sizeof(half)*a*b*slice);

	for(int i = 0;i<p;i++){
		hipMemcpyAsync(d_Xtemp,X+i*a*b*slice,sizeof(dt)*a*b*slice,hipMemcpyHostToDevice,0);
		f2h(d_Xtemp,h_Xtemp,a*b*slice);
		//hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,a,b*slice,&alpha,d_Xtemp,a,d_Xtemp,a,&beta1,d_X1_X1,a);
		hipblasGemmEx(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,
                           a,a,b*slice,
                           &alpha,h_Xtemp,HIP_R_16F,a,
                           h_Xtemp,HIP_R_16F,a,
                           &beta1,d_X1_X1,HIP_R_32F,a,
                           HIP_R_32F,
                           CUBLAS_GEMM_DEFAULT_TENSOR_OP);
		
		//hipblasSgemmStridedBatched(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,b,a,a,&alpha,d_Xtemp,a,a*b,d_Idemat,a,0,&beta,d_X2,b,b*a,slice);
		hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_T,HIPBLAS_OP_N,
		             b,a,a,
		             &alpha,h_Xtemp,HIP_R_16F,a,a*b,
		             h_Idemat,HIP_R_16F,a,0,
		             &beta,d_X2,HIP_R_32F,b,a*b,slice,		             
		             HIP_R_32F,
		             CUBLAS_GEMM_DEFAULT_TENSOR_OP);
		f2h(d_X2,h_X2,a*b*slice);
		//hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,b,b,a,&alpha,d_X2,b,d_X2,b,&beta1,d_X2_X2,b);
		hipblasGemmEx(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,
                           b,b,a,
                           &alpha,h_X2,HIP_R_16F,b,
                           h_X2,HIP_R_16F,b,
                           &beta1,d_X2_X2,HIP_R_32F,b,
                           HIP_R_32F,
                           CUBLAS_GEMM_DEFAULT_TENSOR_OP);
		//cout<<"X2"<<endl;printTensor(d_X2_X2,3,3,1);
		for (int j = 0;j<p;j++){
			hipMemcpyAsync(d_Xtemp1,X+j*a*b*slice,sizeof(dt)*a*b*slice,hipMemcpyHostToDevice,0);
			//printTensor(d_Xtemp1,3,3,1);
			f2h(d_Xtemp1,h_Xtemp1,a*b*slice);
			//hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,slice,slice,a*b,&alpha,d_Xtemp1,a*b,d_Xtemp,a*b,&beta,d_X3+(i*p+j)*slice*slice,slice);
			hipblasGemmEx(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,
                           slice,slice,a*b,
                           &alpha,h_Xtemp1,HIP_R_16F,a*b,
                           h_Xtemp,HIP_R_16F,a*b,
                           &beta,d_X3+(i*p+j)*slice*slice,HIP_R_32F,slice,
                           HIP_R_32F,
                           CUBLAS_GEMM_DEFAULT_TENSOR_OP);
			//hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,1,1,a*b,&alpha,d_Xtemp,1,d_Xtemp1,a*b,&beta,d_X3_X3+i*c+j,1);
			hipblasGemmEx(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,
                           1,1,a*b,
                           &alpha,h_Xtemp,HIP_R_16F,1,
                           h_Xtemp1,HIP_R_16F,a*b,
                           &beta,d_X3_X3+i*c+j,HIP_R_32F,1,
                           HIP_R_32F,
                           CUBLAS_GEMM_DEFAULT_TENSOR_OP);
		}// d_X3 is size of slice *c transpose to c*slice
		hipblasSgemmStridedBatched(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,slice,slice,slice,&alpha,d_X3+i*c*slice,slice,slice*slice,d_Idemat3,slice,0,&beta,d_X3T,slice,slice*slice,p);
		hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,c,slice,&alpha,d_X3T,slice,&beta,d_X3_X3+i*c*slice,c,d_X3_X3+i*c*slice,c);
	}

	//cout<<"d_X1_X1 is :"<<endl;printTensor(d_X1_X1,4,4,1);
	//cout<<"d_X2_X2 is :"<<endl;printTensor(d_X2_X2,4,4,1);
	//cout<<"d_X3_X3 is :"<<endl;printTensor(d_X3_X3,4,4,1);

	hipFree(d_Xtemp1);
	hipFree(d_X2);
	hipFree(d_X3);
	hipFree(d_X3T);
	hipFree(d_Idemat3);
	hipFree(d_Idemat);

	hipFree(h_Xtemp1);
	hipFree(h_Idemat);
	hipFree(h_X2);
	hipDeviceSynchronize();
//==============================================================================
	cout<<"leaf node is ready"<<endl;	
	dt *d_Ux5,*d_Ux4,*d_Ux3,*d_Ux2;
	hipMalloc((void**)&d_Ux5,sizeof(dt)*b*k[4]);
	hipMalloc((void**)&d_Ux4,sizeof(dt)*a*k[3]);
	hipMalloc((void**)&d_Ux3,sizeof(dt)*c*k[2]);
	hipMalloc((void**)&d_Ux2,sizeof(dt)*a*b*k[1]);
	hipDeviceSynchronize();
//===============这里不用是evdj的方法，也就是不用batch============================
	eig(d_X1_X1,a,a,cusolverH);
	eig(d_X2_X2,b,b,cusolverH);
	eig(d_X3_X3,c,c,cusolverH);

	hipblasScopy(handle,a*k[3],d_X1_X1+a*(a-k[3]),1,d_Ux4,1); 
	hipblasScopy(handle,b*k[4],d_X2_X2+b*(b-k[4]),1,d_Ux5,1);
	hipblasScopy(handle,c*k[2],d_X3_X3+c*(c-k[2]),1,d_Ux3,1);
	//printTensor(d_Ux4,4,4,1);
//=============================================================================	
	
	float *d_U;
	hipMalloc((void**)&d_U,sizeof(float)*a*b*k[1]);
	hipDeviceSynchronize();
	tsqr_svd_half(X,a,b,c,k[1],d_U,handle,cusolverH);

	dim3 threads(1024,1,1);
	dim3 block0((a*b*k[1]+1024-1)/1024,1,1);
	transmission<<<block0,threads>>>(d_U,d_Ux2,a*b,k[1]);
	hipDeviceSynchronize();
	hipFree(d_U);
	//printTensor(d_Ux2,3,3,1);
//====================================================================
	dt *d_B2,*d_B1;
	hipMalloc((void**)&d_B1,sizeof(dt)*k[1]*k[2]); 
	hipMalloc((void**)&d_B2,sizeof(dt)*k[3]*k[4]*k[1]);

	half *d_Ux5_h,*d_Ux4_h,*d_Ux3_h,*d_Ux2_h;
	hipMalloc((void**)&d_Ux5_h,sizeof(half)*b*k[4]);
	hipMalloc((void**)&d_Ux4_h,sizeof(half)*a*k[3]);
	hipMalloc((void**)&d_Ux3_h,sizeof(half)*c*k[2]);
	hipMalloc((void**)&d_Ux2_h,sizeof(half)*a*b*k[1]);
	f2h(d_Ux2,d_Ux2_h,a*b*k[1]);
	f2h(d_Ux3,d_Ux3_h,c*k[2]);
	f2h(d_Ux4,d_Ux4_h,a*k[3]);
	f2h(d_Ux5,d_Ux5_h,b*k[4]);

	ttm_tensorcore(d_Ux2_h,d_Ux4_h,d_Ux5_h,d_B2,a,b,k[1],k[3],k[4],handle);

	float *d_u1u2;
	hipMalloc((void**)&d_u1u2,sizeof(float)*k[1]*c);
	for(unsigned i = 0; i < p; ++i) {
		hipMemcpyAsync(d_Xtemp,X+i*a*b*slice,sizeof(dt)*a*b*slice,hipMemcpyHostToDevice,0);
		f2h(d_Xtemp,h_Xtemp,a*b*slice);
		//hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,k[1],slice,a*b,&alpha,d_Ux2,a*b,d_Xtemp,a*b,&beta,d_u1u2+i*k[1]*slice,k[1]);		
		hipblasGemmEx(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,
                           k[1],slice,a*b,
                           &alpha,d_Ux2_h,HIP_R_16F,a*b,
                           h_Xtemp,HIP_R_16F,a*b,
                           &beta,d_u1u2+i*k[1]*slice,HIP_R_32F,k[1],
                           HIP_R_32F,
                           CUBLAS_GEMM_DEFAULT_TENSOR_OP);
	}
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,k[1],k[2],c,&alpha,d_u1u2,k[1],d_Ux3,c,&beta,d_B1,k[1]);

	//cout<<"B1 is :"<<endl;printTensor(d_B1,4,4,1);
	hipEventRecord( stop,0);    //记录当前时间
	hipEventSynchronize(start);    //Waits for an event to complete.
	hipEventSynchronize(stop);    //Waits for an event to complete.Record之前的任务
	hipEventElapsedTime(&time_elapsed,start,stop);    //计算时间差
	hipEventDestroy(start);    //destory the event
	hipEventDestroy(stop);
	time_elapsed = time_elapsed/1000;
	cout<<"cost time :"<<time_elapsed<<"s"<<endl;




	hipFree(d_Ux2_h);
	hipFree(d_Ux3_h);
	hipFree(d_Ux4_h);
	hipFree(d_Ux5_h);
	hipFree(d_X1_X1);
	hipFree(d_X2_X2);
	hipFree(d_X3_X3);
	hipFree(d_B1);
	hipFree(d_B2);
	hipFree(d_Ux4);
	hipFree(d_Ux3);
	hipFree(d_Ux2);
	hipFree(d_Ux5);
	hipFree(d_u1u2);
	hipFree(d_Xtemp);
	hipFree(h_Xtemp);
	hipblasDestroy(handle);
	hipsolverDnDestroy(cusolverH);
}