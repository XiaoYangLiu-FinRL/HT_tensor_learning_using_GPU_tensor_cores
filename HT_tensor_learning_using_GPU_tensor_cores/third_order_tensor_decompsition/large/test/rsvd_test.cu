#include<iostream>
#include<fstream>
#include <assert.h>
#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<hipsolver.h>
#include<hiprand/hiprand.h>
#include <hipfft/hipfft.h>
#include <math.h>
#include <stdlib.h>
#include <time.h> 
#include <hip/hip_fp16.h>

using namespace std;
void printTensor(float *d_des,long m,long n,long l){
    float *des = new float[m*n*l]();
    hipMemcpy(des,d_des,sizeof(float)*m*n*l,hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    for(int k = 0;k<l;k++){
        for(int i = 0;i<n;i++){
            for(int j = 0;j<m;j++){
                cout<<des[k*m*n+i*m+j]<<" ";
            }
            cout<<endl;
        }
        cout<<"~~~~~~~~~~~~~~~~"<<endl;
    }
    delete[] des;des=nullptr;

}
__global__ void transmission(float *d_A,float *d_B,long a,long b)
{
  long long i = blockIdx.x*blockDim.x+threadIdx.x;
    const long long temp = blockDim.x*gridDim.x;
    while(i<a*b)
    {
      long col=i/a+1;
      long row=i%a;
      d_B[a*(b-col)+row]=d_A[i];
      i+=temp;
    }
 __syncthreads();
}
void QR(float *d_A,int m,int n,hipsolverHandle_t cusolverH)
{
     float *d_work = NULL, *d_tau = NULL;
    int *devInfo = NULL;
    int  lwork = 0; 
    int info_gpu = 0;
    hipMalloc((void**)&d_tau, sizeof(float)*n);
    hipMalloc ((void**)&devInfo, sizeof(int));
    hipsolverDnSgeqrf_bufferSize(cusolverH, m, n, d_A, m, &lwork);
    hipMalloc((void**)&d_work, sizeof(float)*lwork);
    hipsolverDnSgeqrf(cusolverH, m, n, d_A, m, d_tau, d_work, lwork, devInfo);
    hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    hipsolverDnSorgqr(cusolverH,m,n,n,d_A,m,d_tau, d_work,lwork,devInfo);

    if (d_work) hipFree(d_work); d_work = NULL;
    if (devInfo) hipFree(devInfo); devInfo = NULL;
    if (d_tau) hipFree(d_tau); d_tau = NULL;
}
void svd(float *d_B,int m,int n,float *d_UT,float *d_S,float *d_V,hipblasHandle_t cublasH,hipsolverHandle_t cusolverH)
{
    float *d_BT = NULL, *d_U = NULL;
    float *d_work = NULL, *d_rwork = NULL;
    int *devInfo = NULL;
    int lwork = 0,  info_gpu = 0;

    float alpha = 1.0;
    float beta = 0.0;

    hipMalloc((void**)&d_BT, sizeof(float)*m*n);
    hipMalloc((void**)&d_U, sizeof(float)*m*m);
    hipMalloc ((void**)&devInfo, sizeof(int));

    hipblasSgeam(cublasH,HIPBLAS_OP_T, HIPBLAS_OP_N, n, m,&alpha,d_B, m,&beta,d_B, n,d_BT, n);

    hipsolverDnSgesvd_bufferSize(cusolverH,n,m,&lwork );
    hipMalloc((void**)&d_work , sizeof(float)*lwork);
    signed char jobu = 'S'; // all m columns of U
    signed char jobvt = 'S'; // all n columns of VT
    hipsolverDnSgesvd(cusolverH,jobu,jobvt,
        n, m,d_BT,n,d_S,d_V,n,  // ldu
        d_U,m, // ldvt,
        d_work,lwork,d_rwork,devInfo);

    hipblasSgeam(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N,  m, m,&alpha, d_U, m,&beta,d_U, m,d_UT, m);

    if(d_BT) hipFree(d_BT);
    if(d_U) hipFree(d_U); 
    if(d_work) hipFree(d_work);
    if(devInfo) hipFree(devInfo);
    if(d_rwork) hipFree(d_rwork); 

}
void rsvd(float *d_A,float *d_U,int m,int n,int ks,hipblasHandle_t handle,hipsolverHandle_t cusolverH)
{
    int p=20;
    float alpha = 1.0;
    float beta =0.0;
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    float *d_B,*d_C;
    hipMalloc((void**)&d_B, sizeof(float)*n*ks);
    hipMalloc((void**)&d_C,sizeof(float)*m*ks);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
    hiprandGenerateNormal(gen, d_B, n*ks, 0, 1);

    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N, m, ks, n,&alpha,d_A,m,d_B,n,&beta,d_C,m);

    QR(d_C,m,ks,cusolverH);
    for(int i=0;i<p;i++)
    {
        hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,n, ks, m,&alpha,d_A,m,d_C,m,&beta,d_B,n);
        QR(d_B,n,ks,cusolverH);
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, ks, n,&alpha,d_A, m, d_B, n,&beta,d_C,m);
        QR(d_C,m,ks,cusolverH);
    }
    hipblasSgemm(handle,HIPBLAS_OP_T,  HIPBLAS_OP_N,ks, n, m,&alpha,d_C, m, d_A,  m,&beta,d_B, ks);
     float *d_UT,*d_S,*d_V;
    hipMalloc((void**)&d_UT, sizeof(float)*ks*ks);
    hipMalloc((void**)&d_S,sizeof(float)*ks);
    hipMalloc((void**)&d_V,sizeof(float)*n*n);

    svd(d_B,ks,n,d_UT, d_S, d_V,handle,cusolverH);
    hipblasSgemm(handle,HIPBLAS_OP_N, HIPBLAS_OP_N,m, ks, ks,&alpha,d_C, m,d_UT, ks,&beta,d_U, m);

    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_UT);
    hipFree(d_S);
    hipFree(d_V);
}
int main()
{
	int m = 10000;
    int n = 10000;
    float alpha = 1.0;
    float beta = 0.0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);  
    hipsolverHandle_t cusolverH = NULL;
    hipsolverDnCreate(&cusolverH);
    float *A;
    hipHostAlloc((void**)&A,sizeof(float)*m*n, 0);
    printf("init data\n");
    for(long long  i = 0; i < m*n; ++i) {
        /* code */
        A[i]= rand()*0.1/(RAND_MAX*0.1);;
    }
     int ks = 1000; 
	float *d_A,*d_AAT,*d_U,*d_U2;
	hipMalloc((void**)&d_A,sizeof(float)*m*n);
    hipMalloc((void**)&d_AAT,sizeof(float)*m*n);
    hipMalloc((void**)&d_U,sizeof(float)*m*ks);
    hipMalloc((void**)&d_U2,sizeof(float)*m*ks);
    hipMemcpyAsync(d_A,A,sizeof(float)*m*n,hipMemcpyHostToDevice,0);
    hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,n,m,&alpha,d_A,m,&beta,d_A,m,d_AAT,n);
   

    rsvd(d_A,d_U,m,n,ks,handle,cusolverH);
    printTensor(d_U,4,4,1);
    hipDeviceSynchronize();
    rsvd(d_AAT,d_U2,n,m,ks,handle,cusolverH);
    printTensor(d_U2,4,4,1);
    
}