#include<iostream>
#include<fstream>
#include <assert.h>
#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<hipsolver.h>
#include<hiprand/hiprand.h>
#include <hipfft/hipfft.h>
#include <math.h>
#include <stdlib.h>
#include <time.h> 
#include <hip/hip_fp16.h>
typedef float dt;
using namespace std;

void  gentuTensor1(dt *X,long a,long b,long c,long r1,long r2,long r3)
{
    dt *A,*B,*C,*G;
    hipHostAlloc((void**)&A,sizeof(dt)*a*r1,0);
    hipHostAlloc((void**)&B, sizeof(dt)*b*r2,0);
    hipHostAlloc((void**)&C, sizeof(dt)*c*r3,0);
    hipHostAlloc((void**)&G, sizeof(dt)*r1*r2*r3,0);
    srand(123);

    for(long long i=0;i<a*r1;i++)
    {
        A[i] = rand()*0.1/(RAND_MAX*0.1);
    }
    for(long long i=0;i<b*r2;i++)
    {
        B[i] = rand()*0.1/(RAND_MAX*0.1);
    }
    for(long long i=0;i<c*r3;i++)
    {
        C[i] = rand()*0.1/(RAND_MAX*0.1);
    }
    for(long long i=0;i<r1*r2*r3;i++)
    {
        G[i] = rand()*0.1/(RAND_MAX*0.1);
    }

    dt * d_A,*d_B,*d_C,*d_X,*d_G;
    hipMalloc((void**)&d_A,sizeof(dt)*a*r1);
    hipMalloc((void**)&d_B,sizeof(dt)*b*r2);
    hipMalloc((void**)&d_C,sizeof(dt)*c*r3);
    hipMalloc((void**)&d_X,sizeof(dt)*a*b*c);
    hipMalloc((void**)&d_G,sizeof(dt)*r2*r1*r3);

    hipMemcpyAsync(d_A, A,sizeof(dt)*a*r1,hipMemcpyHostToDevice,0);
    hipMemcpyAsync(d_B, B,sizeof(dt)*b*r2,hipMemcpyHostToDevice,0);
    hipMemcpyAsync(d_C, C,sizeof(dt)*c*r3,hipMemcpyHostToDevice,0);
    hipMemcpyAsync(d_G, G,sizeof(dt)*r1*r2*r3,hipMemcpyHostToDevice,0);
    dt *d_AG,*d_AGB;
    hipMalloc((void**)&d_AG,sizeof(dt)*a*r2*r3);
    hipMalloc((void**)&d_AGB,sizeof(dt)*b*a*r3);
    dt alpha = 1.0;
    dt beta =0.0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,a,r2*r3,r1,&alpha,d_A,a,d_G,r1,&beta,d_AG,a);
    hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,b,r2,&alpha,d_AG,a,a*r2,d_B,b,0,&beta,d_AGB,a,a*b,r3);
    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a*b,c,r3,&alpha,d_AGB,a*b,d_C,c,&beta,d_X,a*b);
    hipMemcpyAsync(X,d_X,sizeof(dt)*a*b*c,hipMemcpyDeviceToHost,  0);
    hipDeviceSynchronize();
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_G);
    hipFree(d_AGB);
    hipFree(d_AG);
    hipFree(d_X);
    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);
    hipHostFree(G);
    hipblasDestroy(handle);
}
void printTensor(float *d_des,long m,long n,long l){
    float *des = new float[m*n*l]();
    hipMemcpy(des,d_des,sizeof(float)*m*n*l,hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    for(int k = 0;k<l;k++){
        for(int i = 0;i<n;i++){
            for(int j = 0;j<m;j++){
                cout<<des[k*m*n+i*m+j]<<" ";
            }
            cout<<endl;
        }
        cout<<"~~~~~~~~~~~~~~~~"<<endl;
    }
    delete[] des;des=nullptr;

}

__global__ void transmission(dt *d_A,dt *d_B,long a,long b)
{
  long long i = blockIdx.x*blockDim.x+threadIdx.x;
    const long long temp = blockDim.x*gridDim.x;
    while(i<a*b)
    {
      long col=i/a+1;
      long row=i%a;
      d_B[a*(b-col)+row]=d_A[i];
      i+=temp;
    }
 __syncthreads();
}
__global__ void sqrt_gpu2(float *d_A,float *d_B,int b,int k)
{
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    const int temp = blockDim.x*gridDim.x;

    while(i<k*k)
    {
        int row = i%k;
        int col = i/k;
        if(row == col)
        {
            d_B[i] = 1.0/(sqrt(d_A[b-k+row]));     
                
        }
        else
        {
            d_B[i]=0;
        }
        
        i+=temp;
    }
     __syncthreads();
}
__global__ void fuHao(float*d_A,float *d_B,int m,int n)
{
	int i = blockIdx.x*blockDim.x+threadIdx.x;
    const int temp = blockDim.x*gridDim.x;
    while(i<m*n)
    {
    	if(d_A[i] *d_B[i] <0)
    	{
    		d_B[i] = -d_B[i];
    	}

        i+=temp;
    }
     __syncthreads();
}



int main()
{
	int n =200;
	int a = n*n;
	int b = n;
	int k = n*0.1;

	float *A = new float[a*b];
	float *A_mode3 = new float[a*b];

	//gentuTensor1(A,n,n,n,k,k,k);

	for(long i = 0; i < a*b; ++i) {        
        A[i]=rand()*0.1/(RAND_MAX);
    }

	for(int i=0;i<a;i++)
    {
        for(int j = 0; j < b; ++j) {
            A_mode3[i*b+j] = A[i+j*a];          
        }
    }

	float *d_A;
	hipMalloc((void**)&d_A,sizeof(float)*a*b);
    hipMemcpy(d_A,A,sizeof(float)*a*b,hipMemcpyHostToDevice);

    hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipsolverHandle_t cusolverH = NULL;
	hipsolverDnCreate(&cusolverH);
	float alpha = 1.0;
	float beta =0.0;

	float *d_ATA;
	hipMalloc((void**)&d_ATA, sizeof(float)*b*b);

	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,b,b,a,&alpha,d_A,a,d_A,a,&beta,d_ATA,b);

	dim3 threads(1024,1,1);
    dim3 block0((k*k+1024-1)/1024,1,1);

    float *d_W = NULL;
    int *devInfo = NULL;
    float *d_work = NULL;
    int  lwork = 0;

    hipMalloc ((void**)&d_W, sizeof(float) * b);
    hipMalloc ((void**)&devInfo, sizeof(int));

    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    hipsolverDnSsyevd_bufferSize(
        cusolverH,jobz,uplo,b,d_ATA,b,d_W,&lwork);
     hipMalloc((void**)&d_work, sizeof(float)*lwork);
     hipsolverDnSsyevd(
        cusolverH,jobz,uplo,b,d_ATA,b,
        d_W,d_work,lwork,devInfo);

    float *d_S,*d_ST,*d_AK,*d_U,*d_U_t;
    hipMalloc((void**)&d_S,sizeof(float)*k*k);
    hipMalloc((void**)&d_ST,sizeof(float)*b*k);
    hipMalloc((void**)&d_AK,sizeof(float)*b*k); //b行k列
    hipMalloc((void**)&d_U,sizeof(float)*a*k);
    hipMalloc((void**)&d_U_t,sizeof(float)*a*k);

     sqrt_gpu2<<<threads,block0>>>(d_W,d_S,b,k);
     
	hipblasScopy(handle,b*k,d_ATA+b*(b-k),1,d_AK,1); //后k列

	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,b,k,k,&alpha,d_AK,b,d_S,k,&beta,d_ST,b);
	//cublasSgemm(handle,CUBLAS_OP_N,CUBLAS_OP_N,a,k,b,&alpha,d_A,a,d_ST,b,&beta,d_U,a);
	//对d_A分片乘法来做

	int slice2 = a/5;
	float *d_Amode3,*d_tempB,*d_UT;
	hipMalloc((void**)&d_Amode3,sizeof(float)*b*slice2);
	hipMalloc((void**)&d_tempB,sizeof(float)*k*slice2);
	hipMalloc((void**)&d_UT,sizeof(float)*a*k);


	 for(int i = 0; i < 5; ++i) {

        hipMemcpy(d_Amode3,A_mode3+i*b*slice2,sizeof(float)*b*slice2,hipMemcpyHostToDevice);
        //printTensor(d_Amode3,4,4,1);
        hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,k,slice2,b,
                &alpha,d_ST,b,d_Amode3,b,
                &beta,d_tempB,k
                );
                
        hipblasScopy(handle,k*slice2,d_tempB,1,d_U+i*k*slice2,1);
        hipDeviceSynchronize();
    }
    hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,a,k,&alpha,d_U,k,&beta,d_U,k,d_UT,a);
    dim3 block3((a*k+1024-1)/1024,1,1);
	transmission<<<1024,block3>>>(d_UT,d_U_t,a,k);
	hipDeviceSynchronize();

	

	printTensor(d_U_t,4,4,1);



	//直接svd

	int *devInfo3 = NULL;
    float *d_work3 = NULL;
    float *d_rwork3 = NULL;
    int lwork3 = 0;
    int info_gpu3 = 0;
    float *d_S2 = NULL;
    float *d_U2 = NULL;
    float *d_VT = NULL;

    hipMalloc ((void**)&d_S2  , sizeof(float)*b);
    hipMalloc ((void**)&d_U2  , sizeof(float)*a*a);
    hipMalloc ((void**)&d_VT , sizeof(float)*a*b);
    hipMalloc ((void**)&devInfo3, sizeof(int));

    hipsolverDnSgesvd_bufferSize(cusolverH,a,b,&lwork3 );
    hipMalloc((void**)&d_work3 , sizeof(float)*lwork3);
    signed char jobu = 'S'; // all m columns of U
    signed char jobvt = 'S'; // all n columns of VT

    hipsolverDnSgesvd (cusolverH,jobu,jobvt,
        a,b,d_A,a,
        d_S2,
        d_U2,
        a,  // ldu
        d_VT,
        a, // ldvt,
        d_work3,
        lwork3,
        d_rwork3,
        devInfo3);
   printTensor(d_U2,4,4,1);

   fuHao<<<1024,block3>>>(d_U2,d_U_t,a,k);

    float alpha1=-1.0;
    float re=0.0;
    float before = 0.0;

    hipblasSaxpy(handle,a*k,&alpha1,d_U2,1,d_U_t,1);
    //printTensor(d_Ux2,a,b,k); 
    hipblasSnrm2(handle,a*k,d_U_t,1,&re);
    hipblasSnrm2(handle,a*k,d_U2,1,&before);
    hipDeviceSynchronize();
    cout<<"error rate "<<re/before<<endl;


    // 结果，在保证了两者不存在相反数的情况下，在40000*200 的时候  误差达到了0.1

}