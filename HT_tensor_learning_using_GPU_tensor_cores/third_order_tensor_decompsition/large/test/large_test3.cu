#include<iostream>
#include<fstream>
#include <assert.h>
#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<hipsolver.h>
#include<hiprand.h>
#include <hipfft/hipfft.h>
#include <math.h>
#include <stdlib.h>
#include <time.h> 
#include <hip/hip_fp16.h>

typedef float dt;
using namespace std;
oid  gentuTensor1(float *X,long a,long b,long c,long r1,long r2,long r3)
{
    dt *A,*B,*C,*G;
    hipHostAlloc((void**)&A,sizeof(dt)*a*r1,0);
    hipHostAlloc((void**)&B, sizeof(dt)*b*r2,0);
    hipHostAlloc((void**)&C, sizeof(dt)*c*r3,0);
    hipHostAlloc((void**)&G, sizeof(dt)*r1*r2*r3,0);
    srand(123);

    for(long long i=0;i<a*r1;i++)
    {
        A[i] = rand()*0.1/(RAND_MAX*0.1);
    }
    for(long long i=0;i<b*r2;i++)
    {
        B[i] = rand()*0.1/(RAND_MAX*0.1);
    }
    for(long long i=0;i<c*r3;i++)
    {
        C[i] = rand()*0.1/(RAND_MAX*0.1);
    }
    for(long long i=0;i<r1*r2*r3;i++)
    {
        G[i] = rand()*0.1/(RAND_MAX*0.1);
    }
    dt * d_A,*d_B,*d_C,*d_X,*d_G;
    hipMalloc((void**)&d_A,sizeof(dt)*a*r1);
    hipMalloc((void**)&d_B,sizeof(dt)*b*r2);
    hipMalloc((void**)&d_C,sizeof(dt)*c*r3);
    hipMalloc((void**)&d_X,sizeof(dt)*a*b*c);
    hipMalloc((void**)&d_G,sizeof(dt)*r2*r1*r3);

    hipMemcpyAsync(d_A, A,sizeof(dt)*a*r1,hipMemcpyHostToDevice,0);
    hipMemcpyAsync(d_B, B,sizeof(dt)*b*r2,hipMemcpyHostToDevice,0);
    hipMemcpyAsync(d_C, C,sizeof(dt)*c*r3,hipMemcpyHostToDevice,0);
    hipMemcpyAsync(d_G, G,sizeof(dt)*r1*r2*r3,hipMemcpyHostToDevice,0);
    dt *d_AG,*d_AGB;
    hipMalloc((void**)&d_AG,sizeof(dt)*a*r2*r3);
    hipMalloc((void**)&d_AGB,sizeof(dt)*b*a*r3);
    dt alpha = 1.0;
    dt beta =0.0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,a,r2*r3,r1,&alpha,d_A,a,d_G,r1,&beta,d_AG,a);
    hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,b,r2,&alpha,d_AG,a,a*r2,d_B,b,0,&beta,d_AGB,a,a*b,r3);  
    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a*b,c,r3,&alpha,d_AGB,a*b,d_C,c,&beta,d_X,a*b);
    hipMemcpyAsync(X,d_X,sizeof(dt)*a*b*c,hipMemcpyDeviceToHost,  0);
    hipDeviceSynchronize();
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_G);
    hipFree(d_AGB);
    hipFree(d_AG);
    hipFree(d_X);
    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);
    hipHostFree(G);
    hipblasDestroy(handle);
}

int main()
{	
	int a = 100;
    int b = 100;
    int c = 100;
    int k = 10;

    float *A = new float[a*b*c];
    float *A_mode3 = new float[a*b*c];

    gentuTensor1(A,a,b,c,k,k,k);
    
}