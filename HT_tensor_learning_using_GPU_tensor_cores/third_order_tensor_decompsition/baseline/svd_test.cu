#include<iostream>
#include<fstream>
#include <assert.h>
#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<hipsolver.h>
#include<hiprand/hiprand.h>
#include <hipfft/hipfft.h>
#include <math.h>
#include <stdlib.h>
#include <time.h> 
#include <hip/hip_fp16.h>
using namespace std;


__global__ void upper(float *A,float *R,int m,int n)
{
	long long i = blockIdx.x*blockDim.x+threadIdx.x;
	const long long temp = blockDim.x*gridDim.x;

	 while(i<n*n)
	{	
		long row=i/n;
		long col=i%n;
		if(row>=col) 	
			R[i]=A[row*m+col];
		else
			R[i]=0;
		i+=temp;		
	}
	__syncthreads();
}


void printTensor(float *d_des,long m,long n,long l){
	float *des = new float[m*n*l]();
	hipMemcpy(des,d_des,sizeof(float)*m*n*l,hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	for(long k = 0;k<l;k++){
		for(long i = 0;i<n;i++){
			for(long j = 0;j<m;j++){
				cout<<des[k*m*n+i*m+j]<<" ";
			}
			cout<<endl;
		}
		cout<<"~~~~~~~~~~~~~~~~"<<endl;
	}
	delete[] des;des=nullptr;
}
void qr_svd_2(float *d_A,float *d_U,int a,int b)  //这里 a <= b
{

     float *d_upper;    
    hipMalloc((void**)&d_upper, sizeof(float)*a*a);

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  float alpha = 1.0;
  float beta = 0.0;
  hipsolverHandle_t cusolverH = NULL;
  hipsolverDnCreate(&cusolverH);

  float *d_AT;
  hipMalloc((void**)&d_AT,sizeof(float)*a*b);

  hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,b,a,
              &alpha,d_A,a,&beta,d_A,a,d_AT,b
              );

    float *TAU;
    int *devInfo=NULL;
    int lwork_geqrf = 0;

    float *d_work=NULL;
    float *d_work2=NULL;
    int lwork2 = 0;

    dim3 threads(1024,1,1);
    dim3 block0((a*b+1024-1)/1024,1,1);

    hipMalloc((void**)&TAU, sizeof(float)*a);
    hipMalloc ((void**)&devInfo, sizeof(int));
  hipsolverDnSgeqrf_bufferSize(cusolverH,b,a,d_AT,b,&lwork_geqrf);
  hipMalloc((void**)&d_work, sizeof(float)*lwork_geqrf);
    hipsolverDnSgeqrf(cusolverH,
                     b,a,
                     d_AT,b,
                     TAU,
                     d_work,
                     lwork_geqrf,
                     devInfo
                     );
    hipDeviceSynchronize();
    upper<<<block0,threads>>>(d_AT,d_upper,b,a); //R  a*a
    hipDeviceSynchronize();
    hipFree(d_AT);
    float *d_upperT;
    hipMalloc((void**)&d_upperT,sizeof(float)*a*a);
    hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,a,a,
                &alpha,d_upper,a,&beta,d_upper,a,d_upperT,a
                );


    float *d_W;
   hipMalloc((void**)&d_W,sizeof(float)*a);
    float *d_RR_V;
    hipMalloc((void**)&d_RR_V,sizeof(float)*a*a);
    //SVD
  signed char jobu = 'A'; // all m columns of U
    signed char jobvt = 'N';
    float *d_rwork=NULL;
  hipsolverDnSgesvd_bufferSize(cusolverH,
                              a,a,&lwork2
                              );
  hipMalloc((void**)&d_work2,sizeof(float)*lwork2);
  hipsolverDnSgesvd (
        cusolverH,
        jobu,
        jobvt,
        a,
        a,
        d_upperT,
        a,
        d_W,
        d_U,
        a,  // ldu
        d_RR_V,
        a, // ldvt,
        d_work2,
        lwork2,
        d_rwork,
        devInfo);
  hipDeviceSynchronize();
     cout<<"~~~~~~"<<endl;printTensor(d_U,4,4,1);
    hipFree(d_A);
     hipFree(d_W);
    hipFree(TAU);
    hipFree(d_RR_V);
    hipFree(d_upper);
    hipFree(d_upperT);
    hipFree(d_work);
    hipFree(d_work2);
    hipFree(devInfo);
    hipsolverDnDestroy(cusolverH);
    hipblasDestroy(handle);
}

void gesvdj(float *d_AT,float *d_V,int b,int a)
 //需要对 d_AT做SVD，然后求出d_V
{
	int m = b,n=a;
    
    float *d_U;
   // int *devInfo = NULL;
    float *d_work = NULL;
    //float *d_rwork = NULL;
    float *d_S=NULL;
    int *d_info = NULL; 
    //float *d_W = NULL;  // W = S*VT
    int lwork = 0;
    int info = 0; 

    hipsolverHandle_t cusolverH;
    hipsolverDnCreate(&cusolverH);
     hipStream_t stream = NULL;
     hipsolverGesvdjInfo_t gesvdj_params = NULL;
     float tol = 1.e-7;
     int max_sweeps = 15;
     hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
     hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
     hipsolverSetStream(cusolverH, stream);
      hipsolverDnCreateGesvdjInfo(&gesvdj_params);

      int econ = 1;

    hipMalloc ((void**)&d_S  , sizeof(float)*n);
    //cudaMalloc ((void**)&d_U  , sizeof(float)*m*m);
    hipMalloc ((void**)&d_U , sizeof(float)*m*m);
    hipMalloc ((void**)&d_info, sizeof(int));
    //cudaMalloc ((void**)&d_W  , sizeof(float)*m*n);

   hipsolverDnXgesvdjSetTolerance(
        gesvdj_params,
        tol);

   hipsolverDnXgesvdjSetMaxSweeps(
        gesvdj_params,
        max_sweeps);

   hipsolverDnSgesvdj_bufferSize(
        cusolverH,
        jobz, /* CUSOLVER_EIG_MODE_NOVECTOR: compute singular values only */
              /* CUSOLVER_EIG_MODE_VECTOR: compute singular value and singular vectors */
        econ, /* econ = 1 for economy size */
        m,    /* nubmer of rows of A, 0 <= m */
        n,    /* number of columns of A, 0 <= n  */
        d_AT,  /* m-by-n */
        m,  /* leading dimension of A */
        d_S,  /* min(m,n) */
              /* the singular values in descending order */
        d_U,  /* m-by-m if econ = 0 */
              /* m-by-min(m,n) if econ = 1 */
        m,  /* leading dimension of U, ldu >= max(1,m) */
        d_V,  /* n-by-n if econ = 0  */
              /* n-by-min(m,n) if econ = 1  */
        m,  /* leading dimension of V, ldv >= max(1,n) */
        &lwork,
        gesvdj_params);
    hipMalloc((void**)&d_work , sizeof(float)*lwork);

   hipsolverDnSgesvdj(
        cusolverH,
        jobz,  /* CUSOLVER_EIG_MODE_NOVECTOR: compute singular values only */
               /* CUSOLVER_EIG_MODE_VECTOR: compute singular value and singular vectors */
        econ,  /* econ = 1 for economy size */
        m,     /* nubmer of rows of A, 0 <= m */
        n,     /* number of columns of A, 0 <= n  */
        d_AT,   /* m-by-n */
        m,   /* leading dimension of A */
        d_S,   /* min(m,n)  */               /* the singular values in descending order */
        d_U,   /* m-by-m if econ = 0 */          
        m,   /* leading dimension of U, ldu >= max(1,m) */
        d_V,   /* n-by-n if econ = 0  */               /* n-by-min(m,n) if econ = 1  */
        n,   /* leading dimension of V, ldv >= max(1,n) */
        d_work,
        lwork,
        d_info,
        gesvdj_params);
hipDeviceSynchronize();
hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
 if ( 0 == info ){
        printf("gesvdj converges \n");
    }else if ( 0 > info ){
        printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }else{
        printf("WARNING: info = %d : gesvdj does not converge \n", info );
    }

    if (d_S    ) hipFree(d_S);
    if (d_V    ) hipFree(d_V);
    if (d_info) hipFree(d_info);
    if (d_work ) hipFree(d_work);

    if (cusolverH) hipsolverDnDestroy(cusolverH);
    if (stream      ) hipStreamDestroy(stream);
    if (gesvdj_params) hipsolverDnDestroyGesvdjInfo(gesvdj_params);
}



int main()
{
	int a = 200;
	int b = 400;

	float* A = new float[a*b];
	for(long i=0;i<a*b;i++)
   {
        A[i]=rand()*1.0/(RAND_MAX*1.0);
   }
   //   用QR分解之后再SVD
   float* d_A;
   float* d_U;
   hipMalloc((void**)&d_U,sizeof(float)*a*a);
   hipMalloc((void**)&d_A,sizeof(float)*a*b);
   hipMemcpy(d_A,A,sizeof(float)*a*b,hipMemcpyHostToDevice);
   qr_svd_2(d_A,d_U,a,b);
   printTensor(d_U,3,3,1);
   // 对d_A取转置，然后取SVD之后的V，再把V转置
   hipblasHandle_t handle;
   hipblasCreate(&handle);
   float alpha = 1.0;
   float beta = 0.0;

   float* d_AT,*d_V,*d_VT;
   hipMalloc((void**)&d_AT,sizeof(float)*a*b);
   hipMalloc((void**)&d_V,sizeof(float)*a*a);
   hipMalloc((void**)&d_VT,sizeof(float)*a*a);
   hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,b,a,&alpha,d_A,a,&beta,d_A,a,d_AT,b);
   gesvdj(d_AT,d_V,b,a);
   hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,a,a,&alpha,d_V,a,&beta,d_V,a,d_VT,a);
   printTensor(d_VT,3,3,1);

}



