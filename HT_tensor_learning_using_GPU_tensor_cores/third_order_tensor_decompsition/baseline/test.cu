#include "hip/hip_runtime.h"
#include<iostream>
#include<fstream>
#include <assert.h>
#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<hipsolver.h>
#include<hiprand.h>
#include "function.h"
#include <hipfft/hipfft.h>
#include <math.h>
#include <stdlib.h>
#include <time.h> 
#include <hip/hip_fp16.h>
using namespace std;
__global__ void mode2(float *A,float *B,long m,long n,long r)
{
  long long i = blockIdx.x*blockDim.x+threadIdx.x;
  long long temp = blockDim.x*gridDim.x;
  __shared__ float temp2[8];  

  while(i<m*r*n){
    
    long long row=i/n;
    long long col = i%n;
    long long ge = i/(m*n);
    temp2[i]=A[(row-ge*m)+(col*m+ge*m*n)];
    B[i]=temp2[i]; 
    i+=temp;
  }
  __syncthreads();
}
void printvec(float *d_des,long m,long n,long l)
{
  float *des = new float[m*n*l]();
  hipMemcpy(des,d_des,sizeof(float)*m*n*l,hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for(int i = 0; i < m*n*l; ++i) {
    cout<<des[i]<<" ";
  }
  cout<<endl;
  cout<<"~~~~~~~~~~~~~~~~"<<endl;
  delete[] des;des=nullptr;
}
int main()
{
	int a=1200;
	int b=1200*1200;
	int c=1200;
  float *A = new float[a*b];
  for(long i = 0;i<a*b;i++){
    A[i] = i;   
  }
  float *d_A,*d_C;
  hipMalloc((void**)&d_A,sizeof(float)*a*b);
  hipMalloc((void**)&d_C,sizeof(float)*a*b);
  hipMemcpy(d_A,A,sizeof(float)*a*b,hipMemcpyHostToDevice);

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  float alpha = 1.0;
  float beta = 0.0;

  hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,b,a,
              &alpha,d_A,a,&beta,d_A,a,d_C,b
              );

/*	
  float *A = new float[a*b*c]();
	for(int i = 0;i<a*b*c;i++){
		A[i] = i;		
	}
	float *d_A;
	hipMalloc((void**)&d_A,sizeof(float)*a*b*c);
	hipMemcpy(d_A,A,sizeof(float)*a*b*c,hipMemcpyHostToDevice);
	float *d_A2,*d_A3,*d_A1;
	hipMalloc((void**)&d_A2,sizeof(float)*a*b*c);
	hipMalloc((void**)&d_A1,sizeof(float)*a*b*c);
	hipMalloc((void**)&d_A3,sizeof(float)*a*b*c);

	mode2<<<128,512>>>(d_A,d_A1,a,b,c);
	hipDeviceSynchronize();


	printvec(d_A,a,b,c);
	printvec(d_A1,a,b,c);*/



	
}