#include "hip/hip_runtime.h"
#include "head.h"

void printTensor(dt *d_des,long m,long n,long l){
	dt *des = new dt[m*n*l]();
	hipMemcpy(des,d_des,sizeof(dt)*m*n*l,hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	for(long k = 0;k<l;k++){
		for(long i = 0;i<n;i++){
			for(long j = 0;j<m;j++){
				cout<<des[k*m*n+i*m+j]<<" ";
			}
			cout<<endl;
		}
		cout<<"~~~~~~~~~~~~~~~~"<<endl;
	}
	delete[] des;des=nullptr;
}
void printvec(float *d_des,long m,long n,long l)
{
  float *des = new float[m*n*l]();
  hipMemcpy(des,d_des,sizeof(float)*m*n*l,hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for(int i = 0; i < m*n*l; ++i) {
    cout<<des[i]<<" ";
  }
  cout<<endl;
  cout<<"~~~~~~~~~~~~~~~~"<<endl;
  delete[] des;des=nullptr;
}
__global__  void floattohalf(dt *AA,half *BB,long m){
  long i = blockIdx.x*blockDim.x+threadIdx.x;
  const long temp = blockDim.x*gridDim.x;
  if(i<m){
    BB[i]=__float2half(AA[i]);
    i+=temp;
  }
  __syncthreads();
}

void f2h(dt *A,half *B,long num){
  dim3 threads(512,1,1);
  dim3 blocks((num+512-1)/512,1,1); 
  floattohalf<<<blocks,threads>>>(A,B,num);
}
__global__ void tensorToMode1(dt *T1,dt *T2,int m,int n,int k ){
  long i = blockIdx.x*blockDim.x+threadIdx.x;
  const long temp = blockDim.x*gridDim.x;
  while(i<m*n*k){
    long tube = i/(m*n);
    long row = (i-tube*(m*n))%m;
    long col = (i-tube*(m*n))/m;
    T2[tube*m*n+col*m+row] = T1[tube*m*n+col*m+row];
    i+=temp;
  }
  __syncthreads();
  
}

__global__ void tensorToMode2(dt *T1,dt *T2,int m,int n,int k){
  long i = blockIdx.x*blockDim.x+threadIdx.x;
  const long temp = blockDim.x*gridDim.x;
  while(i<m*n*k){
    long tube = i/(m*n);
    long row = (i-tube*(m*n))%m;
    long col = (i-tube*(m*n))/m;
    T2[tube*m*n+row*n+col] = T1[tube*m*n+col*m+row];
    i+=temp;
  }
    __syncthreads();
}

__global__ void tensorToMode3(dt *T1,dt *T2,int m,int n,int k){
  long i = blockIdx.x*blockDim.x+threadIdx.x;
  const long temp = blockDim.x*gridDim.x;
  while(i<m*n*k){
    long tube = i/(m*n);
    long row = (i-tube*(m*n))%m;
    long col = (i-tube*(m*n))/m;
    T2[k*(col*m+row)+tube] = T1[tube*m*n+col*m+row];
    i+=temp;
  }
    __syncthreads();
}

__global__ void truncate_h(dt *d_A,dt *d_B,long a,long b)
{
  long long i = blockIdx.x*blockDim.x+threadIdx.x;
  const long long temp = blockDim.x*gridDim.x;
  while(i<a*b)
  {
    d_B[i]=(d_A+(a-b)*a)[i];
    i+=temp;
  }
  __syncthreads();
}
__global__ void transmission(dt *d_A,dt *d_B,long a,long b)
{
  long long i = blockIdx.x*blockDim.x+threadIdx.x;
    const long long temp = blockDim.x*gridDim.x;
    while(i<a*b)
    {
      d_B[i]=d_A[i];
      i+=temp;
    }
 __syncthreads();
}

void genHtensor(dt *X,long a,long b,long c)
{	
	srand((unsigned)time(NULL)); 
   int size=a;
   int k[5];
   int q=7;
   int w=3;
   for(int i =0;i<5;i++){
        k[i]=(rand() % (q-w+1))+ w; //3-10随机整数 
        //k[i]=(int)(a*0.1);    
   }
   k[0]=1;
   dt *U5,*U4,*U3,*B2,*B1;
   hipHostAlloc((void**)&U5,sizeof(dt)*size*k[4],0);
   hipHostAlloc((void**)&U4,sizeof(dt)*size*k[3],0);
   hipHostAlloc((void**)&U3,sizeof(dt)*size*k[2],0);
   hipHostAlloc((void**)&B2,sizeof(dt)*k[3]*k[4]*k[1],0);
   hipHostAlloc((void**)&B1,sizeof(dt)*k[1]*k[2]*k[0],0);

   for(long i=0;i<size*k[4];i++)
   {
        U5[i]=rand()*2.0/RAND_MAX - 1.0;
   }
   for(long i=0;i<size*k[3];i++)
   {
        U4[i]=rand()*2.0/RAND_MAX - 1.0;
   }
   for(long i=0;i<size*k[2];i++)
   {
        U3[i]=rand()*2.0/RAND_MAX - 1.0;
   }
   for(long i=0;i<k[3]*k[4]*k[1];i++)
   {
        B2[i]=rand()*2.0/RAND_MAX - 1.0;
   }
   for(long i=0;i<k[1]*k[2]*k[0];i++)
   {
        B1[i]=rand()*2.0/RAND_MAX - 1.0;
   }


   dt *d_U5,*d_U4,*d_U3,*d_B2,*d_B1;
   hipMalloc((void**)&d_U5, sizeof(dt)*size*k[4]);
   hipMalloc((void**)&d_U4,sizeof(dt)*size*k[3]);
   hipMalloc((void**)&d_U3,sizeof(dt)*size*k[2]);
   hipMalloc((void**)&d_B2,sizeof(dt)*k[3]*k[4]*k[1]);
   hipMalloc((void**)&d_B1,sizeof(dt)*k[1]*k[2]*k[0]);

   hipMemcpy(d_U5,U5,sizeof(dt)*size*k[4],hipMemcpyHostToDevice);
   hipMemcpy(d_U4,U4,sizeof(dt)*size*k[3],hipMemcpyHostToDevice);
   hipMemcpy(d_U3,U3,sizeof(dt)*size*k[2],hipMemcpyHostToDevice);
   hipMemcpy(d_B2,B2,sizeof(dt)*k[3]*k[4]*k[1],hipMemcpyHostToDevice);
   hipMemcpy(d_B1,B1,sizeof(dt)*k[1]*k[2]*k[0],hipMemcpyHostToDevice);

   hipblasHandle_t handle;
   hipblasCreate(&handle);
   dt alpha = 1.0;
   dt beta = 0.0;

   dt *d_U2,*d_X;
   hipMalloc((void**)&d_U2,sizeof(dt)*size*size*k[1]);
   hipMalloc((void**)&d_X,sizeof(dt)*size*size*size);

   dt*d_U4B2;
   hipMalloc((void**)&d_U4B2, sizeof(dt)*size*k[4]*k[1]);
   //ttm B2 x1 U4 x2 U5
   hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,
               size,k[4]*k[1],k[3],
               &alpha,d_U4,size,d_B2,k[3],
               &beta,d_U4B2,size
               );
   hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,
                             size,size,k[4],
                             &alpha,d_U4B2,size,size*k[4],d_U5,size,0,
                             &beta,d_U2,size,size*k[4],k[1]
                             );
   //ttm B1 x1 U2 x2 U3
   dt *d_U2B1;
   hipMalloc((void**)&d_U2B1, sizeof(dt)*size*size*k[2]);
   hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,
               size*size,k[2],k[1],
               &alpha,d_U2,size*size,d_B1,k[1],
               &beta,d_U2B1,size*size
               );
   hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,
               size*size,size,k[2],
               &alpha,d_U2B1,size*size,d_U3,size,
               &beta,d_X,size*size
               );
   hipDeviceSynchronize();
   hipMemcpy(X,d_X,sizeof(dt)*size*size*size,hipMemcpyDeviceToHost);
   

   hipHostFree(U5);
   hipHostFree(U4);
   hipHostFree(U3);
   hipHostFree(B2);
   hipHostFree(B1);

   hipFree(d_U5);
   hipFree(d_U4);
   hipFree(d_U3);
   hipFree(d_B1);
   hipFree(d_B2);
   hipFree(d_U4B2);
   hipFree(d_U2B1);
   hipFree(d_X);
   hipblasDestroy(handle);
}
__global__ void upper(float *A,float *R,int m,int n)
{
	long long i = blockIdx.x*blockDim.x+threadIdx.x;
	const long long temp = blockDim.x*gridDim.x;

	 while(i<n*n)
	{	
		long row=i/n;
		long col=i%n;
		if(row>=col) 	
			R[i]=A[row*m+col];
		else
			R[i]=0;
		i+=temp;		
	}
	__syncthreads();
}
void qr_svd(dt *d_A,dt *d_U,int a,int b)
{

	 float *d_upper;
    
    hipMalloc((void**)&d_upper, sizeof(float)*b*b);
   // hipMalloc((void**)&d_U, sizeof(float)*a*b);

	   float *TAU;
    int *devInfo=NULL;
    int lwork_geqrf = 0;
    int lwork_orgqr = 0;
    int lwork = 0;
    float *d_work=NULL;
    float *d_work2=NULL;
    int lwork2 = 0;

    dim3 threads(1024,1,1);
	dim3 block0((a*b+1024-1)/1024,1,1);

    hipMalloc((void**)&TAU, sizeof(float)*b);
    hipMalloc ((void**)&devInfo, sizeof(int));
    hipsolverHandle_t cusolverH = NULL;
	hipsolverDnCreate(&cusolverH);
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	float alpha = 1.0;
	float beta = 0.0;

	 hipsolverDnSgeqrf_bufferSize(cusolverH,a,b,d_A,a,&lwork_geqrf);
   hipsolverDnSorgqr_bufferSize(cusolverH,
  	                            a,
  	                            b,
  	                            b,
  	                            d_A,
  	                            a,
  	                            TAU,
  	                            &lwork_orgqr);
  	lwork = (lwork_geqrf > lwork_orgqr)? lwork_geqrf : lwork_orgqr;

  	hipMalloc((void**)&d_work, sizeof(float)*lwork);
    hipsolverDnSgeqrf(cusolverH,
                     a,b,
                     d_A,a,
                     TAU,
                     d_work,
                     lwork,
                     devInfo
                     );
    hipDeviceSynchronize();
    upper<<<block0,threads>>>(d_A,d_upper,a,b); //R  b*b

    hipDeviceSynchronize();

    hipsolverDnSorgqr(cusolverH,   // Q a*b
                     a,b,b,d_A,
                     a,
                     TAU,
                     d_work,
                     lwork,
                     devInfo
                     );
    hipDeviceSynchronize();   
	 float *d_W;
	 hipMalloc((void**)&d_W,sizeof(float)*b);
	 float *d_RR;
    hipMalloc((void**)&d_RR,sizeof(float)*b*b);
    float *d_RR_V;
    hipMalloc((void**)&d_RR_V,sizeof(float)*b*b);
    //SVD
	signed char jobu = 'A'; // all m columns of U
    signed char jobvt = 'A';
    float *d_rwork=NULL;
	hipsolverDnSgesvd_bufferSize(cusolverH,
	                            b,b,&lwork2
	                            );
	hipMalloc((void**)&d_work2,sizeof(float)*lwork2);
	hipsolverDnSgesvd (
        cusolverH,
        jobu,
        jobvt,
        b,
        b,
        d_upper,
        b,
        d_W,
        d_RR,
        b,  // ldu
        d_RR_V,
        b, // ldvt,
        d_work2,
        lwork2,
        d_rwork,
        devInfo);
	hipDeviceSynchronize();
	hipFree(d_RR_V);

	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,
	            a,b,b,&alpha,d_A,a,d_RR,b,&beta,d_U,a
	            );


	   hipFree(d_A);
	   hipFree(d_W);
    hipFree(TAU);
    hipFree(d_RR);
    hipFree(d_upper);
    hipFree(d_work);
    hipFree(d_work2);
    hipFree(devInfo);
    hipsolverDnDestroy(cusolverH);
    hipblasDestroy(handle);

}



__global__ void norm_sum(dt *A,dt *B,int a)
{
	long long i = blockIdx.x*blockDim.x+threadIdx.x;
	const long long temp = blockDim.x*gridDim.x;
	while(i<a)
	{
		B[a-i-1]=A[i]*A[i];
		//B[a-i-1]=A[i];
		i=i+temp;
	}
	__syncthreads();

}
__global__ void upper_1(float *R,int n)
{
	long long i = blockIdx.x*blockDim.x+threadIdx.x;
	const long long temp = blockDim.x*gridDim.x;

	 while(i<n*n)
	{	
		long row=i/n;
		long col=i%n;
		if(row>=col) 	
			R[i]=1;
		else
			R[i]=0;
		i+=temp;		
	}
	__syncthreads();
}
__global__ void sqrt_T(dt *A,dt *B,int a)
{
	long long i = blockIdx.x*blockDim.x+threadIdx.x;
	const long long temp = blockDim.x*gridDim.x;

	while(i<a)
	{
		B[a-i-1]=sqrt(A[i]);
		//B[a-i-1]=A[i];
		i+=temp;
	}
 	__syncthreads();
}

void qr_svd_2(dt *d_A,dt *d_U,int a,int b)  //这里 a <= b
{

     float *d_upper;    
    hipMalloc((void**)&d_upper, sizeof(float)*a*a);

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  dt alpha = 1.0;
  dt beta = 0.0;
  hipsolverHandle_t cusolverH = NULL;
  hipsolverDnCreate(&cusolverH);

  dt *d_AT;
  hipMalloc((void**)&d_AT,sizeof(dt)*a*b);

  hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,b,a,
              &alpha,d_A,a,&beta,d_A,a,d_AT,b
              );
    float *TAU;
    int *devInfo=NULL;
    int lwork_geqrf = 0;
    int lwork_orgqr=0;
    int lwork;
    float *d_work=NULL;
    float *d_work2=NULL;
    int lwork2 = 0;

    dim3 threads(1024,1,1);
    dim3 block0((a*b+1024-1)/1024,1,1);

    hipMalloc((void**)&TAU, sizeof(float)*a);
    hipMalloc ((void**)&devInfo, sizeof(int));
    hipsolverDnSgeqrf_bufferSize(cusolverH,b,a,d_AT,b,&lwork_geqrf);

    hipsolverDnSgeqrf_bufferSize(cusolverH,b,a,d_AT,b,&lwork_geqrf);
   hipsolverDnSorgqr_bufferSize(cusolverH,
                                b,
                                a,
                                a,
                                d_AT,
                                b,
                                TAU,
                                &lwork_orgqr);
    lwork = (lwork_geqrf > lwork_orgqr)? lwork_geqrf : lwork_orgqr;

    hipMalloc((void**)&d_work, sizeof(float)*lwork_geqrf);
    hipsolverDnSgeqrf(cusolverH,
                     b,a,
                     d_AT,b,
                     TAU,
                     d_work,
                     lwork_geqrf,
                     devInfo
                     );
    hipDeviceSynchronize();
    upper<<<block0,threads>>>(d_AT,d_upper,b,a); //R  a*a
    hipDeviceSynchronize();
    hipFree(d_AT);

    dt *d_upperT;
    hipMalloc((void**)&d_upperT,sizeof(dt)*a*a);
    hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,a,a,
                &alpha,d_upper,a,&beta,d_upper,a,d_upperT,a
                );


    float *d_W;
   hipMalloc((void**)&d_W,sizeof(float)*a);
    float *d_RR_V;
    hipMalloc((void**)&d_RR_V,sizeof(float)*a*a);
    //SVD
  signed char jobu = 'A'; // all m columns of U
    signed char jobvt = 'A';
    float *d_rwork=NULL;
  hipsolverDnSgesvd_bufferSize(cusolverH,
                              a,a,&lwork2
                              );
  hipMalloc((void**)&d_work2,sizeof(float)*lwork2);
  hipsolverDnSgesvd (
        cusolverH,
        jobu,
        jobvt,
        a,
        a,
        d_upperT,
        a,
        d_W,
        d_U,
        a,  // ldu
        d_RR_V,
        a, // ldvt,
        d_work2,
        lwork2,
        d_rwork,
        devInfo);
  hipDeviceSynchronize();
    
    hipFree(d_A);
     hipFree(d_W);
    hipFree(TAU);
    //hipFree(d_U);
    hipFree(d_RR_V);
    hipFree(d_upper);
    hipFree(d_upperT);
    hipFree(d_work);
    hipFree(d_work2);
    hipFree(devInfo);
    hipsolverDnDestroy(cusolverH);
    hipblasDestroy(handle);
}

void gesvda(dt *d_A,dt *d_U,int a,int b,int k)
{
  //A输入   U输出left  V输出 right

    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;
    const int batchSize = 1;
    const long m = a;
    const int n = b;
    const int lda = m;
    const int ldu = m;
    const int ldv = n;
    const int rank = b;
    const long long int strideA = (long long int)lda*n;
    const long long int strideS = n;
    const long long int strideU = (long long int)ldu*n;
    const long long int strideV = (long long int)ldv*n;
    //float A[strideA*batchSize] = { 1.0, 4.0, 2.0, 2.0, 5.0, 1.0, 10.0, 8.0, 6.0, 9.0, 7.0, 5.0};
   
    float *d_S = NULL;  /* singular values */
  
    float *d_V = NULL;  /* right singular vectors */
    int *d_info = NULL;  /* error info */
    int lwork = 0;       /* size of workspace */
    float *d_work = NULL; /* devie workspace for gesvda */
    const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvectors.
    double RnrmF[batchSize]; /* residual norm */
    int info[batchSize];  /* host copy of error info */

    hipsolverDnCreate(&cusolverH);
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipsolverSetStream(cusolverH, stream);
   
    hipMalloc ((void**)&d_S   , sizeof(float)*strideS*batchSize);
    hipMalloc ((void**)&d_V   , sizeof(float)*strideV*batchSize);
    hipMalloc ((void**)&d_info, sizeof(int)*batchSize);

   

    hipsolverDnSgesvdaStridedBatched_bufferSize(
        cusolverH,
        jobz, /* HIPSOLVER_EIG_MODE_NOVECTOR: compute singular values only */
              /* HIPSOLVER_EIG_MODE_VECTOR: compute singular value and singular vectors */
        rank, /* number of singular values */
        m,    /* nubmer of rows of Aj, 0 <= m */
        n,    /* number of columns of Aj, 0 <= n  */
        d_A,     /* Aj is m-by-n */
        lda,     /* leading dimension of Aj */
        strideA, /* >= lda*n */
        d_S,     /* Sj is rank-by-1, singular values in descending order */
        strideS, /* >= rank */
        d_U,     /* Uj is m-by-rank */
        ldu,     /* leading dimension of Uj, ldu >= max(1,m) */
        strideU, /* >= ldu*rank */
        d_V,     /* Vj is n-by-rank */
        ldv,     /* leading dimension of Vj, ldv >= max(1,n) */
        strideV, /* >= ldv*rank */
        &lwork,
        batchSize /* number of matrices */
    );
    hipMalloc((void**)&d_work , sizeof(float)*lwork);
    hipsolverDnSgesvdaStridedBatched(
        cusolverH,
        jobz, /* HIPSOLVER_EIG_MODE_NOVECTOR: compute singular values only */
              /* HIPSOLVER_EIG_MODE_VECTOR: compute singular value and singular vectors */
        rank, /* number of singular values */
        m,    /* nubmer of rows of Aj, 0 <= m */
        n,    /* number of columns of Aj, 0 <= n  */
        d_A,     /* Aj is m-by-n */
        lda,     /* leading dimension of Aj */
        strideA, /* >= lda*n */
        d_S,     /* Sj is rank-by-1 */
                 /* the singular values in descending order */
        strideS, /* >= rank */
        d_U,     /* Uj is m-by-rank */
        ldu,     /* leading dimension of Uj, ldu >= max(1,m) */
        strideU, /* >= ldu*rank */
        d_V,     /* Vj is n-by-rank */
        ldv,     /* leading dimension of Vj, ldv >= max(1,n) */
        strideV, /* >= ldv*rank */
        d_work,
        lwork,
        d_info,
        RnrmF,
        batchSize /* number of matrices */
    );
    hipDeviceSynchronize();
    hipMemcpy(info, d_info, sizeof(int)*batchSize, hipMemcpyDeviceToHost);

    if ( 0 > info[0] ){
        printf("%d-th parameter is wrong \n", -info[0]);
        exit(1);
    }
    for(int idx = 0 ; idx < batchSize; idx++){
        if ( 0 == info[idx] ){
            printf("%d-th matrix, svda converges \n", idx );
        }else{
           printf("WARNING: info[%d] = %d : svda does not converge \n", idx, info[idx] );
        }
    }

    hipFree(d_S);
    hipFree(d_A);
    hipFree(d_V);
    hipFree(d_info);
    hipFree(d_work);
    hipStreamDestroy(stream);
    hipsolverDnDestroy(cusolverH);
}
// void gesvdj(float *d_AT,float *d_V,int b,int a)
//  //需要对 d_AT做SVD，然后求出d_V
// {
//   int m = b,n=a;
    
//     float *d_U;
//    // int *devInfo = NULL;
//     float *d_work = NULL;
//     //float *d_rwork = NULL;
//     float *d_S=NULL;
//     int *d_info = NULL; 
//     //float *d_W = NULL;  // W = S*VT
//     int lwork = 0;
//     int info = 0; 

//     hipsolverHandle_t cusolverH;
//     hipsolverDnCreate(&cusolverH);
//      hipStream_t stream = NULL;
//      hipsolverGesvdjInfo_t gesvdj_params = NULL;
//      float tol = 1.e-7;
//      int max_sweeps = 15;
//      hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
//      hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
//      hipsolverSetStream(cusolverH, stream);
//       hipsolverDnCreateGesvdjInfo(&gesvdj_params);

//       int econ = 1;

//     hipMalloc ((void**)&d_S  , sizeof(float)*n);
//     //hipMalloc ((void**)&d_U  , sizeof(float)*m*m);
//     hipMalloc ((void**)&d_U , sizeof(float)*m*m);
//     hipMalloc ((void**)&d_info, sizeof(int));
//     //hipMalloc ((void**)&d_W  , sizeof(float)*m*n);

//    hipsolverDnXgesvdjSetTolerance(
//         gesvdj_params,
//         tol);

//    hipsolverDnXgesvdjSetMaxSweeps(
//         gesvdj_params,
//         max_sweeps);

//    hipsolverDnSgesvdj_bufferSize(
//         cusolverH,
//         jobz, /* HIPSOLVER_EIG_MODE_NOVECTOR: compute singular values only */
//               /* HIPSOLVER_EIG_MODE_VECTOR: compute singular value and singular vectors */
//         econ, /* econ = 1 for economy size */
//         m,    /* nubmer of rows of A, 0 <= m */
//         n,    /* number of columns of A, 0 <= n  */
//         d_AT,  /* m-by-n */
//         m,  /* leading dimension of A */
//         d_S,  /* min(m,n) */
//               /* the singular values in descending order */
//         d_U,  /* m-by-m if econ = 0 */
//               /* m-by-min(m,n) if econ = 1 */
//         m,  /* leading dimension of U, ldu >= max(1,m) */
//         d_V,  /* n-by-n if econ = 0  */
//               /* n-by-min(m,n) if econ = 1  */
//         m,  /* leading dimension of V, ldv >= max(1,n) */
//         &lwork,
//         gesvdj_params);
//     hipMalloc((void**)&d_work , sizeof(float)*lwork);

//    hipsolverDnSgesvdj(
//         cusolverH,
//         jobz,  /* HIPSOLVER_EIG_MODE_NOVECTOR: compute singular values only */
//                /* HIPSOLVER_EIG_MODE_VECTOR: compute singular value and singular vectors */
//         econ,  /* econ = 1 for economy size */
//         m,     /* nubmer of rows of A, 0 <= m */
//         n,     /* number of columns of A, 0 <= n  */
//         d_AT,   /* m-by-n */
//         m,   /* leading dimension of A */
//         d_S,   /* min(m,n)  */               /* the singular values in descending order */
//         d_U,   /* m-by-m if econ = 0 */          
//         m,   /* leading dimension of U, ldu >= max(1,m) */
//         d_V,   /* n-by-n if econ = 0  */               /* n-by-min(m,n) if econ = 1  */
//         n,   /* leading dimension of V, ldv >= max(1,n) */
//         d_work,
//         lwork,
//         d_info,
//         gesvdj_params);
// hipDeviceSynchronize();
// hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
//  if ( 0 == info ){
//         printf("gesvdj converges \n");
//     }else if ( 0 > info ){
//         printf("%d-th parameter is wrong \n", -info);
//         exit(1);
//     }else{
//         printf("WARNING: info = %d : gesvdj does not converge \n", info );
//     }

//     if (d_S    ) hipFree(d_S);
//     if (d_V    ) hipFree(d_V);
//     if (d_info) hipFree(d_info);
//     if (d_work ) hipFree(d_work);

//     if (cusolverH) hipsolverDnDestroy(cusolverH);
//     if (stream      ) hipStreamDestroy(stream);
//     if (gesvdj_params) hipsolverDnDestroyGesvdjInfo(gesvdj_params);
// }

// void svd_VT(float *d_A,float *d_VT,int a,int b,hipblasHandle_t handle)
// {
//    //hipblasHandle_t handle;
//    //hipblasCreate(&handle);
//    float alpha = 1.0;
//    float beta = 0.0;

//    float* d_AT,*d_V;
//    hipMalloc((void**)&d_AT,sizeof(float)*a*b);
//    hipMalloc((void**)&d_V,sizeof(float)*a*a);
//    //hipMalloc((void**)&d_VT,sizeof(float)*a*a);
//    hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,b,a,&alpha,d_A,a,&beta,d_A,a,d_AT,b);
//    gesvdj(d_AT,d_V,b,a);
//    hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,a,a,&alpha,d_V,a,&beta,d_V,a,d_VT,a);
//    printTensor(d_VT,3,3,1);

//    hipFree(d_AT);
//    hipFree(d_V);
// }

