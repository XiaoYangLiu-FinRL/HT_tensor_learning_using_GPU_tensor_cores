#include "head.h"
void htd4(dt *X,long a,long b,long c,long d,int *k,hipblasHandle_t handle,hipsolverHandle_t cusolverH)
{

  
	dt *d_X,*d_XT;
	hipMalloc((void**)&d_X,sizeof(dt)*a*b*c*d);
  hipMalloc((void**)&d_XT,sizeof(dt)*a*b*c*d);
	hipMemcpy(d_X,X,sizeof(dt)*a*b*c*d,hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	
	dt alpha = 1.0;
	dt beta = 0.0;
	dt alpha1=-1.0;
	dt re=0.0;
	dt before = 0.0;

  dim3 block0((a*b*c*d+1024-1)/1024,1,1);
  dim3 threads(1024,1,1);

	
  //hipStream_t stream = NULL;
  //hipsolverSyevjInfo_t syevj_params = NULL;

  hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,c*d,a*b,&alpha,d_X,a*b,&beta,d_X,a*b,d_XT,c*d);
  hipDeviceSynchronize();
	// 求 mode 展开
	dt *d_X2,*d_X3;
	hipMalloc((void**)&d_X2,sizeof(dt)*a*b*c*d);  // 这里用流式处理加速
	hipMalloc((void**)&d_X3,sizeof(dt)*a*b*c*d);
	//mode-2  mode-3
  mode2<<<block0,threads>>>(d_X,d_X2,a,b,c*d);
  hipDeviceSynchronize();
  mode2<<<block0,threads>>>(d_X,d_X3,a*b,c,d);
  hipDeviceSynchronize();

    //用于 特征分解的矩阵  d_X * d_XT
    dt *d_X1_X1,*d_X2_X2,*d_X3_X3,*d_X4_X4;
    hipMalloc((void**)&d_X1_X1,sizeof(dt)*a*a);
    hipMalloc((void**)&d_X2_X2,sizeof(dt)*b*b);
    hipMalloc((void**)&d_X3_X3,sizeof(dt)*c*c);
    hipMalloc((void**)&d_X4_X4,sizeof(dt)*d*d);

    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,a,b*c*d,
                &alpha,d_X,a,d_X,a,
                &beta,d_X1_X1,a
                );

    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,b,b,a*c*d,
                &alpha,d_X2,b,d_X2,b,
                &beta,d_X2_X2,b
                );
    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,c,c,a*b*d,
                &alpha,d_X3,c,d_X3,c,
                &beta,d_X3_X3,c
                );
    hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,d,d,a*c*b,
                &alpha,d_X,a*b*c,d_X,a*b*c,
                &beta,d_X4_X4,d
                );
    hipFree(d_X2);
    hipFree(d_X3);

    dt *d_Ux7,*d_Ux6,*d_Ux5,*d_Ux4;
    hipMalloc((void**)&d_Ux7,sizeof(dt)*d*k[6]);
    hipMalloc((void**)&d_Ux6,sizeof(dt)*c*k[5]);
    hipMalloc((void**)&d_Ux5,sizeof(dt)*b*k[4]);
    hipMalloc((void**)&d_Ux4,sizeof(dt)*a*k[3]);

    dt *d_sumXXT;
    hipMalloc((void**)&d_sumXXT,sizeof(dt)*a*a*4);
    hipblasScopy(handle,a*a,d_X1_X1,1,d_sumXXT,1);
    hipblasScopy(handle,b*b,d_X2_X2,1,d_sumXXT+a*a,1);
    hipblasScopy(handle,c*c,d_X3_X3,1,d_sumXXT+2*a*a,1);
    hipblasScopy(handle,d*d,d_X4_X4,1,d_sumXXT+3*a*a,1);
    dt *d_W = NULL; 
    int* d_info = NULL; 
    int lwork = 0; 
    dt *d_work = NULL; 

    /*const dt tol = 1.e-7;
    const int max_sweeps = 300;
    const int sort_eig = 1; 
    const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
    const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
   // hipsolverDnCreate(&cusolverH);
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipsolverSetStream(cusolverH, stream);
    hipsolverDnCreateSyevjInfo(&syevj_params);
    hipsolverDnXsyevjSetTolerance(syevj_params,tol);
    hipsolverDnXsyevjSetMaxSweeps(syevj_params,max_sweeps);
    hipsolverDnXsyevjSetSortEig(syevj_params,sort_eig);
    hipMalloc ((void**)&d_W , sizeof(dt) * a * 4);
    hipMalloc ((void**)&d_info, sizeof(int ) * 4);
    hipsolverDnSsyevjBatched_bufferSize(cusolverH, jobz,uplo,a,d_sumXXT,a,d_W,&lwork,syevj_params,4);
    hipMalloc((void**)&d_work, sizeof(dt)*lwork);     
    hipsolverDnSsyevjBatched(cusolverH,jobz,uplo,a,d_sumXXT,a,d_W, d_work,lwork,d_info,syevj_params,4);
    hipDeviceSynchronize();
    int info;
    hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
    if ( 0 == info ){
    printf(" converges \n");        
    }else if ( 0 > info ){
    printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }else{
        printf("WARNING: info = %d :  does not converge \n", info );
     }
    
    //4 leaf nodes
   hipblasScopy(handle,a*k[3],d_sumXXT+a*(a-k[3]),1,d_Ux4,1);
   hipblasScopy(handle,b*k[4],d_sumXXT+a*a+a*(b-k[4]),1,d_Ux5,1);
   hipblasScopy(handle,c*k[5],d_sumXXT+2*a*a+a*(c-k[5]),1,d_Ux6,1);
   hipblasScopy(handle,d*k[6],d_sumXXT+3*a*a+a*(d-k[6]),1,d_Ux7,1);
   hipDeviceSynchronize();
   hipFree(d_sumXXT);

   if (cusolverH) hipsolverDnDestroy(cusolverH);
   if (stream      ) hipStreamDestroy(stream);
   if (syevj_params) hipsolverDnDestroySyevjInfo(syevj_params);*/
  //cout<<"d_Ux6 is :"<<endl;printTensor(d_Ux6,4,4,1,1);
  //cout<<"d_Ux5 is :"<<endl;printTensor(d_Ux5,4,4,1,1);
  //cout<<"d_Ux4 is :"<<endl;printTensor(d_Ux4,4,4,1,1);
//====================================================================

  eig(d_X1_X1,a,a,cusolverH);
  eig(d_X2_X2,b,b,cusolverH);
  eig(d_X3_X3,c,c,cusolverH);
  eig(d_X4_X4,d,d,cusolverH);
  //cout<<"d_X4x4 is :"<<endl;printTensor(d_X4_X4,4,4,1,1);
  hipblasScopy(handle,a*k[3],d_X1_X1+a*(a-k[3]),1,d_Ux4,1); 
  hipblasScopy(handle,b*k[4],d_X2_X2+b*(b-k[4]),1,d_Ux5,1);
  hipblasScopy(handle,c*k[5],d_X3_X3+c*(c-k[5]),1,d_Ux6,1);
  hipblasScopy(handle,d*k[6],d_X4_X4+d*(d-k[6]),1,d_Ux7,1);
//=======================用rsvd来做======================================
  dt *d_Ux3,*d_Ux2;
  hipMalloc((void**)&d_Ux3,sizeof(dt)*c*d*k[2]);
  hipMalloc((void**)&d_Ux2,sizeof(dt)*a*b*k[1]);
  float *d_U;
  hipMalloc((void**)&d_U,sizeof(float)*a*b*k[1]);
  dim3 block_t((a*b*k[1]+1024-1)/1024,1,1);
  

  rsvd(d_X,d_U,a*b,c*d,k[1],handle,cusolverH);
  transmission<<<block_t,threads>>>(d_U,d_Ux2,a*b,k[1]);
  //printTensor(d_Ux2,4,4,1,1);

  rsvd(d_XT,d_U,c*d,a*b,k[2],handle,cusolverH);
  transmission<<<block_t,threads>>>(d_U,d_Ux3,a*b,k[1]);
  hipDeviceSynchronize();
  
  hipFree(d_U);
  hipFree(d_XT);
//====================================================================
  

    //cout<<"d_Ux7 is :"<<endl;printTensor(d_Ux7,4,4,1,1);
    // 求 B3 = U3 x1 U6T x2 U7T
    dt *d_B3,*d_B2,*d_B1;
    hipMalloc((void**)&d_B2,sizeof(dt)*k[3]*k[4]*k[1]);
    hipMalloc((void**)&d_B3,sizeof(dt)*k[5]*k[6]*k[2]);
    hipMalloc((void**)&d_B1,sizeof(dt)*k[1]*k[2]*k[0]);

    ttm(d_Ux3,d_Ux6,d_Ux7,d_B3,c,d,k[2],k[5],k[6],handle); 
    //printTensor(d_B3,4,4,1,1);
    //求 B2 = U2 x1 U4T x2 U5T
    ttm(d_Ux2,d_Ux4,d_Ux5,d_B2,a,b,k[1],k[3],k[4],handle);
    //printTensor(d_B2,4,4,1,1);
    //求 B1 = d_X(a*b)*(c*d) x1 U2T x2 U3T
    ttm(d_X,d_Ux2,d_Ux3,d_B1,a*b,c*d,k[0],k[1],k[2],handle);
    //printTensor(d_B1,4,4,1,1);


  //recover
    dt *d_U4B2,*d_U2_r;
    hipMalloc((void**)&d_U4B2,sizeof(dt)*a*k[4]*k[1]);
    hipMalloc((void**)&d_U2_r,sizeof(dt)*a*b*k[1]);

    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,
              a,k[4]*k[1],k[3],
              &alpha,d_Ux4,a,d_B2,k[3],
              &beta,d_U4B2,a
                );

    hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,
                              a,b,k[4],
                              &alpha,d_U4B2,a,a*k[4],d_Ux5,b,0,
                              &beta,d_U2_r,a,a*b,k[1]
                              );
    //printTensor(d_U2_r,4,4,1,1);  正常
    dt *d_U6B3,*d_U3_r;
    hipMalloc((void**)&d_U6B3,sizeof(dt)*c*k[6]*k[2]);
    hipMalloc((void**)&d_U3_r,sizeof(dt)*c*d*k[2]);

    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,
              c,k[6]*k[2],k[5],
              &alpha,d_Ux6,c,d_B3,k[5],
              &beta,d_U6B3,c
                );

    hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,
                              c,d,k[6],
                              &alpha,d_U6B3,c,c*k[6],d_Ux7,d,0,
                              &beta,d_U3_r,c,c*d,k[2]
                              );

    //printTensor(d_U3_r,4,4,1,1);  这里都是 0 存疑
    dt *d_U2B1,*d_X_r;
    hipMalloc((void**)&d_U2B1,sizeof(dt)*a*b*k[2]);
    hipMalloc((void**)&d_X_r,sizeof(dt)*a*b*c*d);

     hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,
              a*b,k[2],k[1],
              &alpha,d_U2_r,a*b,d_B1,k[1],&beta,d_U2B1,a*b
              );

    

     hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,
                 a*b,c*d,k[2],
                 &alpha,d_U2B1,a*b,d_U3_r,c*d,
                 &beta,d_X_r,a*b
                 );
    // printTensor(d_X_r,4,4,1,1);

    hipblasSaxpy(handle,a*b*c*d,&alpha1,d_X,1,d_X_r,1); 
    hipDeviceSynchronize();

    hipblasSnrm2(handle,a*b*c*d,d_X_r,1,&re);
    hipblasSnrm2(handle,a*b*c*d,d_X,1,&before);
    hipDeviceSynchronize();
    
    cout<<"error rate "<<re/before<<endl;



    hipFree(d_X1_X1);
    hipFree(d_X2_X2);
    hipFree(d_X3_X3);
    hipFree(d_X4_X4);
    hipFree(d_X);
    hipFree(d_Ux7);
    hipFree(d_Ux6);
    hipFree(d_Ux5);
    hipFree(d_Ux4);
    hipFree(d_Ux3);
    hipFree(d_Ux2);
}