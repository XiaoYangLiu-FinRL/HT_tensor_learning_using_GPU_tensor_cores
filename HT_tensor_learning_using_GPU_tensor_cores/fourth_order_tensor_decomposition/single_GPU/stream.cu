#include<iostream>
#include<fstream>
#include <assert.h>
#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<hipsolver.h>
#include<hiprand/hiprand.h>
#include <hipfft/hipfft.h>
#include <math.h>
#include <stdlib.h>
#include <time.h> 
#include <hip/hip_fp16.h>
using namespace std;

void printTensor(float *d_des,long m,long n,long l,long k){
  float *des = new float[m*n*l*k]();
  hipMemcpy(des,d_des,sizeof(float)*m*n*l*k,hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for(int d=0;d<k;d++){
  for(int c = 0;c<l;c++){
    for(int b = 0;b<n;b++){
      for(int a = 0;a<m;a++){
        cout<<des[d*m*n*l+c*m*n+b*m+a]<<" ";
      }
      cout<<endl;
    }
    cout<<"~~~~~~~~~~~~~~~~~~~~~"<<endl;
  }
}
}

int main()
{
	float *A;
	float *B;
	hipHostAlloc((void**)&A,sizeof(float)*16,0);
	hipHostAlloc((void**)&B,sizeof(float)*16,0);
	for(unsigned i = 0; i < 16; ++i) {
		/* code */
		A[i] = i+1;
		B[i] = i+2;
	}
	float *d_A,*d_B,*d_C,*d_D;
	hipMalloc((void**)&d_A,sizeof(float)*16);
	hipMalloc((void**)&d_B,sizeof(float)*16);
	hipMalloc((void**)&d_C,sizeof(float)*16);
	hipMalloc((void**)&d_D,sizeof(float)*16);		
	hipStream_t stream[2];
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	float alpha=1.0f;                                             
	float beta=0.0f;


	hipMemcpyAsync(d_A,A, sizeof(float)*16,hipMemcpyHostToDevice,0);
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,4,4,4,&alpha,d_A,4,d_A,4,&beta,d_C,4);

	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
for(int j=0;j<10;j++){
	hipStreamCreate(&stream[0]);	
	hipblasSetStream(handle,stream[0]);
	hipMemcpyAsync(d_A,A, sizeof(float)*16,hipMemcpyHostToDevice,stream[0]);
	//cudaMemcpyAsync(d_A,A, sizeof(float)*16,cudaMemcpyHostToDevice,0);
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,4,4,4,&alpha,d_A,4,d_A,4,&beta,d_C,4);
	printTensor(d_C,4,4,1,1);
	hipblasDestroy(handle);

	hipStreamCreate(&stream[1]);
	//cublasCreate(&handle);
	hipblasSetStream(handle,stream[1]);
	hipMemcpyAsync(d_B,B, sizeof(float)*16,hipMemcpyHostToDevice,stream[1]);
	//cudaMemcpyAsync(d_B,B, sizeof(float)*16,cudaMemcpyHostToDevice,0);
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,4,4,4,&alpha,d_B,4,d_B,4,&beta,d_D,4);
	printTensor(d_D,4,4,1,1);
	//cublasDestroy(handle);
	hipStreamSynchronize(0); 
	hipStreamDestroy(stream[0]);
	hipStreamDestroy(stream[1]);
}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
 
	std::cout << "COST TIME : " << elapsedTime/10 << std::endl;

}