#include "hip/hip_runtime.h"
#include "head.h"

void printTensor(dt *d_des,long m,long n,long l,long k){
  dt *des = new dt[m*n*l*k]();
  hipMemcpy(des,d_des,sizeof(dt)*m*n*l*k,hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for(int d=0;d<k;d++){
  for(int c = 0;c<l;c++){
    for(int b = 0;b<n;b++){
      for(int a = 0;a<m;a++){
        cout<<des[d*m*n*l+c*m*n+b*m+a]<<" ";
      }
      cout<<endl;
    }
    cout<<"~~~~~~~~~~~~~~~~~~~~~"<<endl;
  }
}
}

__global__  void floattohalf(dt *AA,half *BB,long m){
  long i = blockIdx.x*blockDim.x+threadIdx.x;
  const long temp = blockDim.x*gridDim.x;
  if(i<m){
    BB[i]=__float2half(AA[i]);
    i+=temp;
  }
  __syncthreads();
}

void f2h(dt *A,half *B,long num){
  dim3 threads(512,1,1);
  dim3 blocks((num+512-1)/512,1,1); 
  floattohalf<<<blocks,threads>>>(A,B,num);
}

__global__ void mode2h(half *A,half *B,long m,long n,long r)
{
  long long i = blockIdx.x*blockDim.x+threadIdx.x;
  const long long temp = blockDim.x*gridDim.x;
  while(i<m*r*n){
    long long row=i/n;
    long long col = i%n;
    long long ge = i/(m*n);
    B[i]=A[(row-ge*m)+(col*m+ge*m*n)];    
    i+=temp;
  }
  __syncthreads();  
}


__global__ void tensorToMode1(dt *T1,dt *T2,int m,int n,int k ){
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  const int temp = blockDim.x*gridDim.x;
  while(i<m*n*k){
    int tube = i/(m*n);
    int row = (i-tube*(m*n))%m;
    int col = (i-tube*(m*n))/m;
    T2[tube*m*n+col*m+row] = T1[tube*m*n+col*m+row];
    i+=temp;
  }
  __syncthreads();
  
}

__global__ void tensorToMode2(dt *T1,dt *T2,int m,int n,int k){
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  const int temp = blockDim.x*gridDim.x;
  while(i<m*n*k){
    int tube = i/(m*n);
    int row = (i-tube*(m*n))%m;
    int col = (i-tube*(m*n))/m;
    T2[tube*m*n+row*n+col] = T1[tube*m*n+col*m+row];
    i+=temp;
  }
    __syncthreads();
}

__global__ void tensorToMode3(dt *T1,dt *T2,int m,int n,int k){
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  const int temp = blockDim.x*gridDim.x;
  while(i<m*n*k){
    int tube = i/(m*n);
    int row = (i-tube*(m*n))%m;
    int col = (i-tube*(m*n))/m;
    T2[k*(col*m+row)+tube] = T1[tube*m*n+col*m+row];
    i+=temp;
  }
    __syncthreads();
}

__global__ void sub(dt *A,dt *B,long a,long b,long c)
{
	long long i = blockIdx.x*blockDim.x+threadIdx.x;
	const long long temp = blockDim.x*gridDim.x;
	while(i<a*b*c)
	{
		B[i] = A[i] - B[i];
		i+=temp;
	}
	__syncthreads();
}

__global__ void initIdeMat(dt *AA,int m){
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  const int temp = blockDim.x*gridDim.x;
  while(i<m*m){
    int row = i%m;
    int col = i/m;
    if(row==col){
      AA[col*m+row] = 1;
    }else{
      AA[col*m+row] = 0;
    }
    i+=temp;
  }
  __syncthreads();
}
__global__ void transmission(dt *d_A,dt *d_B,long a,long b)
{
  long long i = blockIdx.x*blockDim.x+threadIdx.x;
    const long long temp = blockDim.x*gridDim.x;
    while(i<a*b)
    {
      long col=i/a+1;
      long row=i%a;
      d_B[a*(b-col)+row]=d_A[i];
      i+=temp;
    }
 __syncthreads();
}

void genHtensor(dt *X,long a,long b,long c,long d)
{	
	 srand((unsigned)time(NULL)); 
   int size=a;
   int k[7];
   int q=7;
   int w=3;
   for(int i =0;i<7;i++){
        k[i]=(rand() % (q-w+1))+ w; //3-10随机整数
       //k[i]=size;             
   }
   k[0]=1;
   dt *U5,*U4,*U3,*U6,*B3,*B2,*B1;
   hipHostAlloc((void**)&U6,sizeof(dt)*size*k[6],0);
   hipHostAlloc((void**)&U5,sizeof(dt)*size*k[5],0);
   hipHostAlloc((void**)&U4,sizeof(dt)*size*k[4],0);
   hipHostAlloc((void**)&U3,sizeof(dt)*size*k[3],0);
   hipHostAlloc((void**)&B3,sizeof(dt)*k[5]*k[6]*k[2],0);
   hipHostAlloc((void**)&B2,sizeof(dt)*k[3]*k[4]*k[1],0);
   hipHostAlloc((void**)&B1,sizeof(dt)*k[1]*k[2]*k[0],0);

   for(long i=0;i<size*k[6];i++)
   {
        U6[i]=rand()*2.0/RAND_MAX - 1.0;
   }
   for(long i=0;i<size*k[5];i++)
   {
        U5[i]=rand()*2.0/RAND_MAX - 1.0;
   }
   for(long i=0;i<size*k[4];i++)
   {
        U4[i]=rand()*2.0/RAND_MAX - 1.0;
   }
   for(long i=0;i<size*k[3];i++)
   {
        U3[i]=rand()*2.0/RAND_MAX - 1.0;
   }
   for(long i=0;i<k[5]*k[6]*k[2];i++)
   {
        B3[i]=rand()*2.0/RAND_MAX - 1.0;
   }
   for(long i=0;i<k[3]*k[4]*k[1];i++)
   {
        B2[i]=rand()*2.0/RAND_MAX - 1.0;
   }
   for(long i=0;i<k[1]*k[2]*k[0];i++)
   {
        B1[i]=rand()*2.0/RAND_MAX - 1.0;
   }


   dt *d_U6,*d_U5,*d_U4,*d_U3,*d_B3,*d_B2,*d_B1;
   hipMalloc((void**)&d_U6,sizeof(dt)*size*k[6]);
   hipMalloc((void**)&d_U5,sizeof(dt)*size*k[5]);
   hipMalloc((void**)&d_U4,sizeof(dt)*size*k[4]);
   hipMalloc((void**)&d_U3,sizeof(dt)*size*k[3]);
   hipMalloc((void**)&d_B3,sizeof(dt)*k[5]*k[6]*k[2]);
   hipMalloc((void**)&d_B2,sizeof(dt)*k[3]*k[4]*k[1]);
   hipMalloc((void**)&d_B1,sizeof(dt)*k[1]*k[2]*k[0]);

   hipMemcpy(d_U6,U6,sizeof(dt)*size*k[6],hipMemcpyHostToDevice);
   hipMemcpy(d_U5,U5,sizeof(dt)*size*k[5],hipMemcpyHostToDevice);
   hipMemcpy(d_U4,U4,sizeof(dt)*size*k[4],hipMemcpyHostToDevice);
   hipMemcpy(d_U3,U3,sizeof(dt)*size*k[3],hipMemcpyHostToDevice);
   hipMemcpy(d_B3,B3,sizeof(dt)*k[5]*k[6]*k[2],hipMemcpyHostToDevice);
   hipMemcpy(d_B2,B2,sizeof(dt)*k[3]*k[4]*k[1],hipMemcpyHostToDevice);
   hipMemcpy(d_B1,B1,sizeof(dt)*k[1]*k[2]*k[0],hipMemcpyHostToDevice);

   hipblasHandle_t handle;
   hipblasCreate(&handle);
   dt alpha = 1.0;
   dt beta = 0.0;

   dt *d_U2,*d_X,*d_U1;
   hipMalloc((void**)&d_U2,sizeof(dt)*size*size*k[2]);
   hipMalloc((void**)&d_U1,sizeof(dt)*size*size*k[1]);
   hipMalloc((void**)&d_X,sizeof(dt)*size*size*size*size);

   dt*d_U5B3;
   hipMalloc((void**)&d_U5B3, sizeof(dt)*size*k[6]*k[2]);
   //ttm B2 x1 U5 x2 U6
   hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,
               size,k[6]*k[2],k[5],
               &alpha,d_U5,size,d_B3,k[5],
               &beta,d_U5B3,size
               );
   hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,
                             size,size,k[6],
                             &alpha,d_U5B3,size,size*k[6],d_U6,size,0,
                             &beta,d_U2,size,size,k[2]
                             );
   //ttm B1 x1 U3 x2 U4
   dt *d_U3B2;
   hipMalloc((void**)&d_U3B2, sizeof(dt)*size*k[4]*k[1]);
   hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,
               size,k[4]*k[1],k[3],
               &alpha,d_U3,size,d_B2,k[3],
               &beta,d_U3B2,size
               );
   hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,
               size,size,k[4],
               &alpha,d_U3B2,size,size*k[4],d_U4,size,0,
               &beta,d_U1,size,size*size,k[1]
               );
   hipDeviceSynchronize();

   // ttm B1 x1 U1 x2 U2
 dt *d_U1B1;
 hipMalloc((void**)&d_U1B1,sizeof(dt)*size*size*k[2]);
   hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,size*size,k[2],k[1],
               &alpha,d_U1,size*size,d_B1,k[1],
               &beta,d_U1B1,size*size
               );
  hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,size*size,size*size,k[2],
              &alpha,d_U1B1,size*size,d_U2,size*size,
              &beta,d_X,size*size
              );
  hipDeviceSynchronize();
  
  //printTensor(d_X,2,2,2,2);

  hipMemcpy(X,d_X,sizeof(dt)*size*size*size*size,hipMemcpyDeviceToHost);
   hipHostFree(U6);
   hipHostFree(U5);
   hipHostFree(U4);
   hipHostFree(U3);
   hipHostFree(B3);
   hipHostFree(B2);
   hipHostFree(B1);
  
   hipFree(d_U6);
   hipFree(d_U5);
   hipFree(d_U4);
   hipFree(d_U3);
   hipFree(d_B3);
   hipFree(d_B1);
   hipFree(d_B2);
   hipFree(d_U5B3);
   hipFree(d_U3B2);
   hipFree(d_U1B1);
   hipFree(d_X);
   hipblasDestroy(handle);
}
__global__ void upper(dt *A,dt *R,int m,int n)
{
	long long i = blockIdx.x*blockDim.x+threadIdx.x;
	const long long temp = blockDim.x*gridDim.x;

	 while(i<n*n)
	{	
		long row=i/n;
		long col=i%n;
		if(row>=col) 	
			R[i]=A[row*m+col];
		else
			R[i]=0;
		i+=temp;		
	}
	__syncthreads();
}

void eig(float *d_A,int m,int n,hipsolverHandle_t cusolverH)
{
    float *d_W = NULL;
    int *devInfo = NULL;
    float *d_work = NULL;
    int  lwork = 0;

    hipMalloc ((void**)&d_W, sizeof(float) * m);
    hipMalloc ((void**)&devInfo, sizeof(int));
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    hipsolverDnSsyevd_bufferSize(cusolverH,jobz,uplo,m,d_A,m,d_W,&lwork);
    hipMalloc((void**)&d_work, sizeof(float)*lwork);
    hipsolverDnSsyevd(cusolverH,jobz,uplo,m,d_A,m,d_W,d_work,lwork,devInfo);
    hipDeviceSynchronize();
   // cout<<"in the function :"<<endl;printTensor(d_A,4,4,1);
    if (d_W    ) hipFree(d_W);
    if (devInfo) hipFree(devInfo);
    if (d_work ) hipFree(d_work);

}

void QR(float *d_A,int m,int n,hipsolverHandle_t cusolverH)
{
     float *d_work = NULL, *d_tau = NULL;
    int *devInfo = NULL;
    int  lwork = 0; 
    int info_gpu = 0;
    hipMalloc((void**)&d_tau, sizeof(float)*n);
    hipMalloc ((void**)&devInfo, sizeof(int));
    hipsolverDnSgeqrf_bufferSize(cusolverH, m, n, d_A, m, &lwork);
    hipMalloc((void**)&d_work, sizeof(float)*lwork);
    hipsolverDnSgeqrf(cusolverH, m, n, d_A, m, d_tau, d_work, lwork, devInfo);
    hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    hipsolverDnSorgqr(cusolverH,m,n,n,d_A,m,d_tau, d_work,lwork,devInfo);

    if (d_work) hipFree(d_work); d_work = NULL;
    if (devInfo) hipFree(devInfo); devInfo = NULL;
    if (d_tau) hipFree(d_tau); d_tau = NULL;
}
void svd(float *d_B,int m,int n,float *d_UT,float *d_S,float *d_V,hipblasHandle_t cublasH,hipsolverHandle_t cusolverH)
{
    float *d_BT = NULL, *d_U = NULL;
    float *d_work = NULL, *d_rwork = NULL;
    int *devInfo = NULL;
    int lwork = 0;

    float alpha = 1.0;
    float beta = 0.0;

    hipMalloc((void**)&d_BT, sizeof(float)*m*n);
    hipMalloc((void**)&d_U, sizeof(float)*m*m);
    hipMalloc ((void**)&devInfo, sizeof(int));

    hipblasSgeam(cublasH,HIPBLAS_OP_T, HIPBLAS_OP_N, n, m,&alpha,d_B, m,&beta,d_B, n,d_BT, n);

    hipsolverDnSgesvd_bufferSize(cusolverH,n,m,&lwork );
    hipMalloc((void**)&d_work , sizeof(float)*lwork);
    signed char jobu = 'S'; // all m columns of U
    signed char jobvt = 'S'; // all n columns of VT
    hipsolverDnSgesvd(cusolverH,jobu,jobvt,
        n, m,d_BT,n,d_S,d_V,n,  // ldu
        d_U,m, // ldvt,
        d_work,lwork,d_rwork,devInfo);

    hipblasSgeam(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N,  m, m,&alpha, d_U, m,&beta,d_U, m,d_UT, m);

    if(d_BT) hipFree(d_BT);
    if(d_U) hipFree(d_U); 
    if(d_work) hipFree(d_work);
    if(devInfo) hipFree(devInfo);
    if(d_rwork) hipFree(d_rwork); 

}
void rsvd(float *d_A,float *d_U,int m,int n,int ks,hipblasHandle_t handle,hipsolverHandle_t cusolverH)
{
    int p=20;
    float alpha = 1.0;
    float beta =0.0;
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    float *d_B,*d_C;
    hipMalloc((void**)&d_B, sizeof(float)*n*ks);
    hipMalloc((void**)&d_C,sizeof(float)*m*ks);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
    hiprandGenerateNormal(gen, d_B, n*ks, 0, 1);
    //printTensor(d_B,3,3,1,1);
    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N, m, ks, n,&alpha,d_A,m,d_B,n,&beta,d_C,m);

    QR(d_C,m,ks,cusolverH);
    for(int i=0;i<p;i++)
    {
        hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,n, ks, m,&alpha,d_A,m,d_C,m,&beta,d_B,n);
        QR(d_B,n,ks,cusolverH);
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, ks, n,&alpha,d_A, m, d_B, n,&beta,d_C,m);
        QR(d_C,m,ks,cusolverH);
    }
    hipblasSgemm(handle,HIPBLAS_OP_T,  HIPBLAS_OP_N,ks, n, m,&alpha,d_C, m, d_A,  m,&beta,d_B, ks);
     float *d_UT,*d_S,*d_V;
    hipMalloc((void**)&d_UT, sizeof(float)*ks*ks);
    hipMalloc((void**)&d_S,sizeof(float)*ks);
    hipMalloc((void**)&d_V,sizeof(float)*n*n);

    svd(d_B,ks,n,d_UT, d_S, d_V,handle,cusolverH);
    hipblasSgemm(handle,HIPBLAS_OP_N, HIPBLAS_OP_N,m, ks, ks,&alpha,d_C, m,d_UT, ks,&beta,d_U, m);

    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_UT);
    hipFree(d_S);
    hipFree(d_V);
}

void qr_svd_2(dt *d_A,dt *d_U,int a,int b)  //这里 a <= b
{

     float *d_upper;    
    hipMalloc((void**)&d_upper, sizeof(float)*a*a);

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  dt alpha = 1.0;
  dt beta = 0.0;
  hipsolverHandle_t cusolverH = NULL;
  hipsolverDnCreate(&cusolverH);

  dt *d_AT;
  hipMalloc((void**)&d_AT,sizeof(dt)*a*b);

  hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,b,a,
              &alpha,d_A,a,&beta,d_A,a,d_AT,b
              );

    float *TAU;
    int *devInfo=NULL;
    int lwork_geqrf = 0;

    float *d_work=NULL;
    float *d_work2=NULL;
    int lwork2 = 0;

    dim3 threads(1024,1,1);
    dim3 block0((a*b+1024-1)/1024,1,1);

    hipMalloc((void**)&TAU, sizeof(float)*a);
    hipMalloc ((void**)&devInfo, sizeof(int));
  hipsolverDnSgeqrf_bufferSize(cusolverH,b,a,d_AT,b,&lwork_geqrf);
  hipMalloc((void**)&d_work, sizeof(float)*lwork_geqrf);
    hipsolverDnSgeqrf(cusolverH,
                     b,a,
                     d_AT,b,
                     TAU,
                     d_work,
                     lwork_geqrf,
                     devInfo
                     );
    hipDeviceSynchronize();
    upper<<<block0,threads>>>(d_AT,d_upper,b,a); //R  a*a
    hipDeviceSynchronize();
    hipFree(d_AT);

    dt *d_upperT;
    hipMalloc((void**)&d_upperT,sizeof(dt)*a*a);
    hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,a,a,
                &alpha,d_upper,a,&beta,d_upper,a,d_upperT,a
                );


    float *d_W;
   hipMalloc((void**)&d_W,sizeof(float)*a);
    float *d_RR_V;
    hipMalloc((void**)&d_RR_V,sizeof(float)*a*a);
    //SVD
  signed char jobu = 'A'; // all m columns of U
    signed char jobvt = 'N';
    float *d_rwork=NULL;
  hipsolverDnSgesvd_bufferSize(cusolverH,
                              a,a,&lwork2
                              );
  hipMalloc((void**)&d_work2,sizeof(float)*lwork2);
  hipsolverDnSgesvd (
        cusolverH,
        jobu,
        jobvt,
        a,
        a,
        d_upperT,
        a,
        d_W,
        d_U,
        a,  // ldu
        d_RR_V,
        a, // ldvt,
        d_work2,
        lwork2,
        d_rwork,
        devInfo);
  hipDeviceSynchronize();
    
    hipFree(d_A);
     hipFree(d_W);
    hipFree(TAU);
    //hipFree(d_U);
    hipFree(d_RR_V);
    hipFree(d_upper);
    hipFree(d_upperT);
    hipFree(d_work);
    hipFree(d_work2);
    hipFree(devInfo);
    hipsolverDnDestroy(cusolverH);
    hipblasDestroy(handle);
}