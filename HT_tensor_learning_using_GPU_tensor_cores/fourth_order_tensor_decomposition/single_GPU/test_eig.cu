#include "hip/hip_runtime.h"
#include<iostream>
#include<fstream>
#include <assert.h>
#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<hipsolver.h>
#include<hiprand.h>
#include "function.h"
#include <hipfft/hipfft.h>
#include <math.h>
#include <stdlib.h>
#include <time.h> 
#include <hip/hip_fp16.h>
using namespace std;

typedef float dt;

void printvec(dt *d_des,long m,long n,long l)
{
  dt *des = new dt[m*n*l]();
  hipMemcpy(des,d_des,sizeof(dt)*m*n*l,hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for(int i = 0; i < m*n*l; ++i) {
    cout<<des[i]<<" ";
  }
  cout<<endl;
  cout<<"~~~~~~~~~~~~~~~~"<<endl;
  delete[] des;des=nullptr;
}


int main()
{
	hipsolverHandle_t cusolverH = NULL;
	hipsolverDnCreate(&cusolverH);
	hipStream_t stream = NULL;
    hipsolverSyevjInfo_t syevj_params = NULL;
	int n=1200;
	dt *A = new dt[n*n];
	for(unsigned i = 0; i < n*n; ++i) {
		/* code */
		A[i] = rand()*1.0/(RAND_MAX*1.0);
	}
	dt *d_A,*d_W;
	hipMalloc((void**)&d_A,sizeof(dt)*n*n);
	hipMalloc ((void**)&d_W, sizeof(float) * n);
	hipMemcpy(d_A, A, sizeof(float) * n * n, hipMemcpyHostToDevice);

	int *devInfo = NULL;
	float *d_work = NULL;
	int  lwork = 0;
/*
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
*/
	clock_t t1,t2;
	double times=0.0;

	t1=clock();	
	const float tol = 1.e-7;
    const int max_sweeps = 15;
	hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
    hipblasFillMode_t  uplo = HIPBLAS_FILL_MODE_LOWER;
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipsolverSetStream(cusolverH, stream);
    hipsolverDnCreateSyevjInfo(&syevj_params);
    hipsolverDnXsyevjSetTolerance(
        syevj_params,
        tol);
    hipsolverDnXsyevjSetMaxSweeps(
        syevj_params,
        max_sweeps);
 
     hipsolverDnSsyevj_bufferSize(
        cusolverH,
        jobz,
        uplo, 
        n,
        d_A,
        n,
        d_W, 
        &lwork,
        syevj_params);
    hipMalloc((void**)&d_work, sizeof(double)*lwork);
    hipsolverDnSsyevj(
        cusolverH,
        jobz,
        uplo, 
        n,
        d_A,
        n,
        d_W, 
        d_work,
        lwork,
        devInfo,
        syevj_params);
/*
    hipDeviceSynchronize();
    hipEventRecord(stop,0);
	float costtime;
	hipEventElapsedTime(&costtime,start,stop);*/

	t2=clock();
		times = (double)(t2-t1)/CLOCKS_PER_SEC;
		
		cout<<"cost time :"<<times<<"s"<<endl;

    printvec(d_W,6,2,1);




}