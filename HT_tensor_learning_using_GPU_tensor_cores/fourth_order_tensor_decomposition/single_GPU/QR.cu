void QR(float *d_A,int m,int n,hipsolverHandle_t cusolverH)
{
    float *d_work = NULL, *d_tau = NULL;
    int *devInfo = NULL;
     int lwork_geqrf = 0;
    int lwork_orgqr = 0;
    int lwork = 0;
    int info_gpu = 0;
    hipMalloc((void**)&d_tau, sizeof(float)*n);
    hipMalloc ((void**)&devInfo, sizeof(int));

    hipsolverDnSgeqrf_bufferSize(
        cusolverH,
        m,
        n,
        d_A,
        m,
        &lwork_geqrf);
    hipsolverDnSorgqr_bufferSize(
        cusolverH,
        m,
        n,
        n,
        d_A,
        m,
        d_tau,
        &lwork_orgqr);
    lwork = (lwork_geqrf > lwork_orgqr)? lwork_geqrf : lwork_orgqr;
    hipMalloc((void**)&d_work, sizeof(double)*lwork);
    hipsolverDnSgeqrf(
        cusolverH,
        m,
        n,
        d_A,
        m,
        d_tau,
        d_work,
        lwork,
        devInfo);
    hipsolverDnSorgqr(
        cusolverH,
        m,
        n,
        n,
        d_A,
        m,
        d_tau,
        d_work,
        lwork,
        devInfo);

    hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
     //printf("after geqrf: info_gpu = %d\n", info_gpu);

    if (d_work) hipFree(d_work); d_work = NULL;
    if (devInfo) hipFree(devInfo); devInfo = NULL;
    if (d_tau) hipFree(d_tau); d_tau = NULL;
}