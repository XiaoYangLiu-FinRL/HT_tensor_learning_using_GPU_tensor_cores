#include "hip/hip_runtime.h"
#include "head.h"

void random_isometry(float *iso,int m,int n) 
{
	
	float *d_A;  //需要生成的A 大小为 n*m  不是 m*n
	hipMalloc((void**)&d_A,sizeof(float)*m*n);
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen,1233ULL);
	hiprandGenerateNormal(gen,d_A,m*n,0,1);
	// 对d_A QR分解，然后得到 Q，R
	hipsolverHandle_t cusolverH = NULL;
	hipsolverDnCreate(&cusolverH);
	int *devInfo2 = NULL;
    float *d_work2 = NULL;
    int  lwork_geqrf = 0;
    int  lwork_orgqr = 0;
    int  lwork2 = 0;
    float *d_R;
    float *d_tau = NULL;
    hipMalloc((void**)&d_R,sizeof(float)*m*m);     
    hipMalloc ((void**)&d_tau, sizeof(float) * m);
    hipMalloc ((void**)&devInfo2, sizeof(int));

    hipsolverDnSgeqrf_bufferSize(cusolverH,n,m,d_A,n,&lwork_geqrf);
    hipsolverDnSorgqr_bufferSize(cusolverH,n,m,m,d_A,n, d_tau,&lwork_orgqr);

    lwork2 = (lwork_geqrf > lwork_orgqr)? lwork_geqrf : lwork_orgqr;
    hipMalloc((void**)&d_work2, sizeof(float)*lwork2);

    hipsolverDnSgeqrf(cusolverH,n,m,d_A,n,d_tau,d_work2,lwork2,devInfo2);
    upper<<<512,512>>>(d_A,d_R,n,m); // 获得R
    hipDeviceSynchronize();
    hipsolverDnSorgqr(cusolverH,n,m,m,d_A,n,d_tau,d_work2,lwork2,devInfo2);
    //d_A 是Q
    float *d_r;
    hipMalloc((void**)&d_r,sizeof(float)*m);

    dim3 block0((m*m+1024-1)/1024,1,1);
    diag_part<<<block0,1024>>>(d_R,d_r,m);

    R_div_r<<<block0,1024>>>(d_R,d_r,m);

    hipblasHandle_t handle;
	hipblasCreate(&handle);
	float alpha = 1.0;
	float beta = 0.0;

	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,m,n,m,&alpha,d_R,m,d_A,n,&beta,iso,m);
	hipDeviceSynchronize();
	hipFree(d_A);
	hipFree(d_R);
	hipFree(d_tau);
	hipFree(d_r);
	hipFree(devInfo2);
	hipblasDestroy(handle);
	hipsolverDnDestroy(cusolverH);

}
void reshape(float *iso,int a,int b,int c)
{	
	//b=c
	float *d_AT,*d_tmp;
	hipMalloc((void**)&d_AT,sizeof(float)*a*b*c);
	hipMalloc((void**)&d_tmp,sizeof(float)*b*b);

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	float alpha = 1.0;
	float beta = 0.0;

	hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,b*c,a,&alpha,iso,a,&beta,iso,a,d_AT,b*c);
	dim3 block0((b*b+1024-1)/1024,1,1);
	initIdeMat<<<block0,1024>>>(d_tmp,b);

	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_T,HIPBLAS_OP_T, b,b,b,&alpha,d_AT, b,b*b,d_tmp, b,0,
                                  &beta,iso, b*b,b,a);
	hipDeviceSynchronize();

	hipFree(d_AT);
	hipFree(d_tmp);
	hipblasDestroy(handle);
}

void random_tree_tn_uniform(int *Ds,float **iso,int **sizeiso)
{
	int *Ds2 = new int[num_layers+1];
	memcpy(Ds2,Ds,sizeof(int)*num_layers);
	Ds2[num_layers] = 1;//top_ranks
	int tmp;
	cout<<" "<<endl;
	for(int j=0;j<num_layers;j++)
	{	
		hipHostAlloc((void**)&sizeiso[j],sizeof(int)*3,0);
		tmp = pow(Ds2[j],2);
		if(Ds2[j+1] == tmp)
		{
			//iso[j]=一个对角阵
			hipMalloc((void**)&iso[j],sizeof(float)*Ds2[j+1]*Ds2[j+1]);
			dim3 block0((Ds2[j+1]*Ds2[j+1]+1024-1)/1024,1,1);
			eye<<<block0,1024>>>(iso[j],Ds2[j+1]);
		}
		else{
			//随机生成
			hipMalloc((void**)&iso[j],sizeof(float)*Ds2[j+1]*tmp);
			random_isometry(iso[j],Ds2[j+1],tmp);
		}
		//reshape(Ds2[j+1],Ds2[j],Ds2[j]) 每列作为一个矩阵，并转置
		
		reshape(iso[j],Ds2[j+1],Ds2[j],Ds2[j]); //reshape之后的size为Ds2[j] * Ds2[j] * Ds2[j+1]
		//这里获得每个iso 的shape
		sizeiso[j][0] = Ds2[j];sizeiso[j][1] = Ds2[j];sizeiso[j][2] = Ds2[j+1];
		//cout<<sizeiso[j][0]<<" "<<sizeiso[j][1]<<" "<<sizeiso[j][2]<<endl;  
	}
	//printTensor(iso[1],Ds2[1],Ds2[1],Ds2[2]);
}