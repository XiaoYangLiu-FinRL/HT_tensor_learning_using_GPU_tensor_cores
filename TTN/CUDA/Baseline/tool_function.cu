#include "hip/hip_runtime.h"
#include "head.h"

void printTensor(float *d_des,long m,long n,long l){
	float *des = new float[m*n*l]();
	hipMemcpy(des,d_des,sizeof(float)*m*n*l,hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	for(long k = 0;k<l;k++){
		for(long i = 0;i<n;i++){
			for(long j = 0;j<m;j++){
				cout<<des[k*m*n+i*m+j]<<" ";
			}
			cout<<endl;
		}
		cout<<"~~~~~~~~~~~~~~~~"<<endl;
	}
	delete[] des;des=nullptr;
}
__global__ void initIdeMat(float *AA,int m){
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  const int temp = blockDim.x*gridDim.x;
  while(i<m*m){
    int row = i%m;
    int col = i/m;
    if(row==col){
      AA[col*m+row] = 1;
    }else{
      AA[col*m+row] = 0;
    }
    i+=temp;
  }
  __syncthreads();
}
__global__ void diag_part(float *d_R,float *d_r,int m)
{
	int i = blockIdx.x*blockDim.x+threadIdx.x;
  const int temp = blockDim.x*gridDim.x;
  while(i<m*m)
  {
  	int row = i%m;
    int col = i/m;
    if(row==col){
      d_r[row]=fabs(d_R[col*m+row]);
    }
    i+=temp;
  }
   __syncthreads();
}

__global__ void R_div_r(float *d_R,float *d_r,int m)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  const int temp = blockDim.x*gridDim.x;
  while(i<m*m)
  {
  	int row = i%m;
    int col = i/m;
    if(row==col){
      d_R[col*m+row] = d_R[col*m+row] / d_r[row];
    }else
    {
    	d_R[col*m+row] = 0;
    }
    i+=temp;
  }
   __syncthreads();
}
__global__ void shift_ham(float *h,int n,float ham_shift)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  const int temp = blockDim.x*gridDim.x;
  while(i<n*n){
    int row = i%n;
    int col = i/n;
    if(row==col){
      h[col*n+row] = h[col*n+row] -ham_shift;
    }
    i+=temp;
  }
  __syncthreads();
}

__global__ void eye(float *AA,int m)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  const int temp = blockDim.x*gridDim.x;
  while(i<m*m){
    int row = i%m;
    int col = i/m;
    if(row==col){
      AA[col*m+row] = 1;
    }else{
      AA[col*m+row] = 0;
    }
    i+=temp;
  }
  __syncthreads();
}
__global__ void upper(float *A,float *R,int m,int n)
{
    long long i = blockIdx.x*blockDim.x+threadIdx.x;
    const long long temp = blockDim.x*gridDim.x;

     while(i<n*n)
    {   
        long row=i/n;
        long col=i%n;
        if(row>=col)    
            R[i]=A[row*m+col];
        else
            R[i]=0;
        i+=temp;        
    }
    __syncthreads();
}

__global__ void mode2(float *A,float *B,long m,long n,long r)
{
  long long i = blockIdx.x*blockDim.x+threadIdx.x;
  const long long temp = blockDim.x*gridDim.x;
  while(i<m*r*n){
    long long row=i/n;
    long long col = i%n;
    long long ge = i/(m*n);
    B[i]=A[(row-ge*m)+(col*m+ge*m*n)];    
    i+=temp;
  }
  __syncthreads();  
}

void ncon_1(float *A_d,float *B_d,float *C_d,vector<int> modeA,vector<int> modeB,vector<int> modeC,unordered_map<int, int64_t> extent,hiptensorHandle_t handle)
{
    typedef float floatTypeA;
    typedef float floatTypeB;
    typedef float floatTypeC;
    typedef float floatTypeCompute;

    hipDataType typeA = HIP_R_32F;
    hipDataType typeB = HIP_R_32F;
    hipDataType typeC = HIP_R_32F;
    hiptensorComputeDescriptor_t typeCompute = CUTENSOR_R_MIN_32F;

    floatTypeCompute alpha = (floatTypeCompute)1.1f;
    floatTypeCompute beta  = (floatTypeCompute)0.0f;

    int nmodeA = modeA.size();
    int nmodeB = modeB.size();
    int nmodeC = modeC.size();

   vector<int64_t> extentC;
    for (auto mode : modeC)
        extentC.push_back(extent[mode]);
    vector<int64_t> extentA;
    for (auto mode : modeA)
        extentA.push_back(extent[mode]);
    vector<int64_t> extentB;
    for (auto mode : modeB)
        extentB.push_back(extent[mode]);

    size_t elementsA = 1;
    for (auto mode : modeA)
        elementsA *= extent[mode];
    size_t elementsB = 1;
    for (auto mode : modeB)
        elementsB *= extent[mode];
    size_t elementsC = 1;
    for (auto mode : modeC)
        elementsC *= extent[mode];

    hiptensorTensorDescriptor_t descA;
    hiptensorInitTensorDescriptor(&handle,
                 &descA,
                 nmodeA,
                 extentA.data(),
                 NULL,/*stride*/
                 typeA, HIPTENSOR_OP_IDENTITY);

    hiptensorTensorDescriptor_t descB;
   hiptensorInitTensorDescriptor(&handle,
                 &descB,
                 nmodeB,
                 extentB.data(),
                 NULL,/*stride*/
                 typeB, HIPTENSOR_OP_IDENTITY);

    hiptensorTensorDescriptor_t descC;
   hiptensorInitTensorDescriptor( &handle,
                 &descC,
                 nmodeC,
                 extentC.data(),
                 NULL,/*stride*/
                 typeC, HIPTENSOR_OP_IDENTITY);

     uint32_t alignmentRequirementA;
     cutensorGetAlignmentRequirement(&handle,
                  A_d,
                  &descA,
                  &alignmentRequirementA);

     uint32_t alignmentRequirementB;
     cutensorGetAlignmentRequirement(&handle,
                  B_d,
                  &descB,
                  &alignmentRequirementB);

     uint32_t alignmentRequirementC;
     cutensorGetAlignmentRequirement(&handle,
                  C_d,
                  &descC, 
                  &alignmentRequirementC);

    cutensorContractionDescriptor_t desc;
   cutensorInitContractionDescriptor(&handle, 
                 &desc,
                 &descA, modeA.data(), alignmentRequirementA,
                 &descB, modeB.data(), alignmentRequirementB,
                 &descC, modeC.data(), alignmentRequirementC,
                 &descC, modeC.data(), alignmentRequirementC,
                 typeCompute);

    cutensorContractionFind_t find;
    cutensorInitContractionFind( 
                 &handle, &find, 
                 HIPTENSOR_ALGO_DEFAULT);

    uint64_t worksize = 0;
    cutensorContractionGetWorkspace(&handle,
                 &desc,
                 &find,
                 CUTENSOR_WORKSPACE_RECOMMENDED, &worksize);

    void *work = nullptr;
    if (worksize > 0)
    {
        if (hipSuccess != hipMalloc(&work, worksize))
        {
            work = nullptr;
            worksize = 0;
        }
    } 

    hiptensorContractionPlan_t plan;
    cutensorInitContractionPlan(&handle,
                 &plan,
                 &desc,
                 &find,
                 worksize);

    hiptensorStatus_t err;
    err = hiptensorContraction(&handle,
                                  &plan,
                                  (void*) &alpha, A_d, B_d,
                                  (void*) &beta,  C_d, C_d, 
                                  work, worksize, 0);
        if (err != HIPTENSOR_STATUS_SUCCESS)
        {
            cout<<"over"<<endl; 
            printf("ERROR: %s in line %d\n", hiptensorGetErrorString(err), __LINE__);
        }
    if (work) hipFree(work);
}

void gesvdj(float *d_A,float *d_U,float *d_V,float *d_S,int m,int n,hipsolverHandle_t cusolverH)
{

    float *d_work = NULL;
    int *d_info = NULL; 

    int lwork = 0;
    int info = 0; 

     hipStream_t stream = NULL;
     hipsolverGesvdjInfo_t gesvdj_params = NULL;
     float tol = 1.e-7;
     int max_sweeps = 15;
     hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
     hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
     hipsolverSetStream(cusolverH, stream);
      hipsolverDnCreateGesvdjInfo(&gesvdj_params);

      int econ = 1;

    hipMalloc ((void**)&d_info, sizeof(int));
   
   hipsolverDnXgesvdjSetTolerance(
        gesvdj_params,
        tol);

   hipsolverDnXgesvdjSetMaxSweeps(
        gesvdj_params,
        max_sweeps);

   hipsolverDnSgesvdj_bufferSize(
        cusolverH,
        jobz, /* HIPSOLVER_EIG_MODE_NOVECTOR: compute singular values only */
              /* HIPSOLVER_EIG_MODE_VECTOR: compute singular value and singular vectors */
        econ, /* econ = 1 for economy size */
        m,    /* nubmer of rows of A, 0 <= m */
        n,    /* number of columns of A, 0 <= n  */
        d_A,  /* m-by-n */
        m,  /* leading dimension of A */
        d_S,  /* min(m,n) */
              /* the singular values in descending order */
        d_U,  /* m-by-m if econ = 0 */
              /* m-by-min(m,n) if econ = 1 */
        m,  /* leading dimension of U, ldu >= max(1,m) */
        d_V,  /* n-by-n if econ = 0  */
              /* n-by-min(m,n) if econ = 1  */
        n,  /* leading dimension of V, ldv >= max(1,n) */
        &lwork,
        gesvdj_params);
    hipMalloc((void**)&d_work , sizeof(float)*lwork);

   hipsolverDnSgesvdj(
        cusolverH,
        jobz,  /* HIPSOLVER_EIG_MODE_NOVECTOR: compute singular values only */
               /* HIPSOLVER_EIG_MODE_VECTOR: compute singular value and singular vectors */
        econ,  /* econ = 1 for economy size */
        m,     /* nubmer of rows of A, 0 <= m */
        n,     /* number of columns of A, 0 <= n  */
        d_A,   /* m-by-n */
        m,   /* leading dimension of A */
        d_S,   /* min(m,n)  */               /* the singular values in descending order */
        d_U,   /* m-by-m if econ = 0 */          
        m,   /* leading dimension of U, ldu >= max(1,m) */
        d_V,   /* n-by-n if econ = 0  */               /* n-by-min(m,n) if econ = 1  */
        n,   /* leading dimension of V, ldv >= max(1,n) */
        d_work,
        lwork,
        d_info,
        gesvdj_params);
hipDeviceSynchronize();

    if (d_info) hipFree(d_info);
    if (d_work ) hipFree(d_work);

    if (stream      ) hipStreamDestroy(stream);
    if (gesvdj_params) hipsolverDnDestroyGesvdjInfo(gesvdj_params);
}

__global__ void tensorToMode3(float *T1,float *T2,int m,int n,int k){
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  const int temp = blockDim.x*gridDim.x;
  while(i<m*n*k){
    int tube = i/(m*n);
    int row = (i-tube*(m*n))%m;
    int col = (i-tube*(m*n))/m;
    T2[k*(col*m+row)+tube] = T1[tube*m*n+col*m+row];
    i+=temp;
  }
    __syncthreads();
}
__global__ void tensorToMode1(float *T1,float *T2,int m,int n,int k ){
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  const int temp = blockDim.x*gridDim.x;
  while(i<m*n*k){
    int tube = i/(m*n);
    int row = (i-tube*(m*n))%m;
    int col = (i-tube*(m*n))/m;
    T2[tube*m*n+col*m+row] = T1[tube*m*n+col*m+row];
    i+=temp;
  }
  __syncthreads();  
}